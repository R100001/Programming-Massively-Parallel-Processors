#include "hip/hip_runtime.h"

#include <stdio.h>

#define DEBUG

#define SIZE 1024

__global__
void reduction(float* X, unsigned int n, float* Y){

    __shared__ float partial_sum[SIZE];

    unsigned int tx = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    // Load the elements of the array into the shared memory
    if(i < n) partial_sum[tx] = X[i];
    else partial_sum[tx] = 0;
    if(i + blockDim.x < n) partial_sum[tx] += X[i + blockDim.x];

    // Accumulate the elements using reduction
    for(unsigned int stride = blockDim.x / 2; stride >= 1; stride >>= 1){
        __syncthreads();
        if(tx < stride)
            partial_sum[tx] += partial_sum[tx + stride];
    }

    if(tx == 0) Y[blockIdx.x] = partial_sum[0];
}


int main(int argc, char* argv[]){
    
    float *h_X;
    float *d_X, *d_Y;
    float res;
    unsigned int n, n_blocks;

    if(argc != 2){
        printf("Usage: %s <number of elements>\n", argv[0]);
        return 1;
    }

    n = atoi(argv[1]);

    // Allocate host memory
    h_X = (float*)malloc(n * sizeof(float));

    // Allocate device memory
    hipMalloc((void**)&d_X, n * sizeof(float));
    hipMalloc((void**)&d_Y, ceil((float)n / SIZE) * sizeof(float));

    // Initialize host memory
    srand(time(NULL));
    for(unsigned int i = 0; i < n; i++)
        h_X[i] = rand() / (float)RAND_MAX;

#ifdef DEBUG
    // for(unsigned int i = 0; i < n; i++)
    //     printf("%f ", h_X[i]);
    // printf("\n");
#endif

    // Copy host memory to device memory
    hipMemcpy(d_X, h_X, n * sizeof(float), hipMemcpyHostToDevice);

    // Launching the kernel
    for(; n > 1; n = n_blocks){
        n_blocks = ceil((float)n / (SIZE * 2));
        reduction<<<n_blocks, SIZE>>>(d_X, n, d_Y);

        hipMemcpy(d_X, d_Y, n_blocks * sizeof(float), hipMemcpyDeviceToDevice);
    }

    // Copy device memory to host memory
    hipMemcpy(&res, d_X, sizeof(float), hipMemcpyDeviceToHost);
    printf("%f\n", res);

#ifdef DEBUG
    printf("Sum: %f\n", res);
#endif

    // Free memory
    hipFree(d_X);
    hipFree(d_Y);
    free(h_X);

    return 0;
}
