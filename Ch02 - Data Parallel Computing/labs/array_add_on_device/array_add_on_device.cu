/*/
 *
 *  This is a program to test the performance
 *  of two different kernels, add_v1 and add_v2.
 *  
 *  The difference between the two kernels is that
 *  in add_v1, i is the fastest changing thread dimension,
 *  while in add_v2, j is the fastest changing thread dimension.
 * 
 *  Compile with:
 *      nvcc array_add_on_device.cu
 * 
 *  Run with:
 *     ./a.out
 * 
 *  Check the performance of the two kernels with:
 *      nvprof ./a.out  
 * 
/*/


#include <hip/hip_runtime.h>
#include <cstdio>

#define IDX(row, col, LDA) ((row) * (LDA) + (col))

// computes c(i,j) = a(i,j) + b(i,j)
// In this case i is the fastest changing thread dimension
__global__ void add_v1(int *a, int *b, int *c, int N, int M)
{
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < N && j < M)
    {
        int idx = IDX(i, j, M);
        c[idx] = a[idx] + b[idx];
    }
}

// computes c(i,j) = a(i,j) + b(i,j)
// In this case j is the fastest changing thread dimension
__global__ void add_v2(int *a, int *b, int *c, int N, int M)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N && j < M)
    {
        int idx = IDX(i, j, M);
        c[idx] = a[idx] + b[idx];
    }
}

int main()
{
    int N = 2 * 1024;
    int M = 2 * 1024;
    int *a, *b, *c;
    dim3 threads(32, 32);
    dim3 blocks(N / threads.x, M / threads.y);

    hipMallocManaged(&a, N * M * sizeof(int));
    hipMallocManaged(&b, N * M * sizeof(int));
    hipMallocManaged(&c, N * M * sizeof(int));

    add_v1<<<blocks, threads>>>(a, b, c, N, M);
    add_v2<<<blocks, threads>>>(a, b, c, N, M);

    hipFree(a);
    hipFree(b);
    hipFree(c);

    hipDeviceSynchronize();
    return 0;
}
