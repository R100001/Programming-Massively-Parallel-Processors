#include "hip/hip_runtime.h"
/*/
 * 
 *  A program that converts an input image with 3 channels to a grayscale image with 1 channel
 * 
 *  Compile with:
 *      nvcc image_color_to_grayscale.cu
 * 
 *  Run with:
 *     ./a.out <block_size> <image_width> <image_height>
 * 
/*/

#include <stdio.h>

#define DEBUG

#define NUM_CHANNELS 3

__global__
void color_to_grayscale_conversion(unsigned char* in, unsigned char* out, int width, int height){

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < 0 || row >= height || col < 0 || col >= width) return;

    int grey_offset = row * width + col;

    int rgb_offset = grey_offset * NUM_CHANNELS;

    unsigned char r = in[rgb_offset + 0];
    unsigned char g = in[rgb_offset + 1];
    unsigned char b = in[rgb_offset + 2];

    out[grey_offset] = (unsigned char)(0.21f * r + 0.71f * g + 0.07f * b);
}


int main(int argc, char* argv[]){

    int block_dim;
    int image_width;
    int image_height;
    int size;
    unsigned char* h_input_image, *h_output_image;
    unsigned char* d_input_image, *d_output_image;

    if(argc != 4){
        printf("Usage: %s <block_dim> <image_width> <image_height>\n", argv[0]);
        return 1;
    }
    block_dim = atoi(argv[1]);
    image_width = atoi(argv[2]);
    image_height = atoi(argv[3]);
    size = image_width * image_height;

    // Allocate memory for the input and output images on host
    h_input_image = (unsigned char*) malloc(NUM_CHANNELS * size * sizeof(unsigned char));
    h_output_image = (unsigned char*) malloc(size * sizeof(unsigned char));
    
    // Allocate memory for the input and output images on device
    hipMalloc((void**) &d_input_image, NUM_CHANNELS * size * sizeof(unsigned char));
    hipMalloc((void**) &d_output_image, size * sizeof(unsigned char));

    // Initialize the input image with random values between 0 and 255
    srand(time(NULL));
    for(int i = 0; i < NUM_CHANNELS * size; ++i)
        h_input_image[i] = rand() % 256;

#ifdef DEBUG
    // Show the input image
    for(int c = 0; c < NUM_CHANNELS; ++c){
        for(int i = 0; i < image_height; ++i){
            for(int j = 0; j < image_width; ++j){
                printf("%3d ", h_input_image[NUM_CHANNELS * (i * image_width + j) + c]);
            }
            printf("\n");
        }
        printf("\n");
    }
    printf("\n");
#endif

    // Copy the input image to the device
    hipMemcpy(d_input_image, h_input_image, NUM_CHANNELS * size * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Blur the image
    dim3 dimBlock(block_dim, block_dim, 1);
    dim3 dimGrid(ceil((float)image_width/block_dim), ceil((float)image_height/block_dim), 1);
    color_to_grayscale_conversion<<<dimGrid, dimBlock>>>(d_input_image, d_output_image, image_width, image_height);

    // Copy the output back to the host
    hipMemcpy(h_output_image, d_output_image, size * sizeof(unsigned char), hipMemcpyDeviceToHost);

#ifdef DEBUG
    // Show the output image
    for(int i = 0; i < image_height; ++i){
        for(int j = 0; j < image_width; ++j)
            printf("%3d ", h_output_image[i * image_width + j]);
        printf("\n");
    }
    printf("\n");
#endif

    // Free the device memory
    hipFree(d_input_image);
    hipFree(d_output_image);

    // Free the host memory
    free(h_input_image);
    free(h_output_image);
}