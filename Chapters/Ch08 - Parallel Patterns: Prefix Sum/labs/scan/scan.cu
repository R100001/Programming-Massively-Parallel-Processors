

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

//#define DEBUG

#define SECTION_SIZE 256
#define SHARED_MEM_SIZE 8192 // 8KB (2^13)

// Exercise 4
// This kernel can handle up to SECTION_SIZE elements
__global__
void Kogge_Stone_exclusive_scan(float *X, float *Y, int n){

    __shared__ float XY[SECTION_SIZE];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;

    // Load elements into shared memory
    XY[tx] = (i < n && tx != 0) ? X[i - 1] : 0;

    // Perform scan operation
    for (unsigned int stride = 1; stride < blockDim.x; stride <<= 1){

        __syncthreads();

        if (tx >= stride) XY[tx] += XY[tx - stride];
    }

    // Write result to global memory
    if (i < n) Y[i] = XY[tx];

}

//----------------------------------------------------------------------


// Exercise 5
// This kernel can handle up to SECTION_SIZE elements
__global__
void Kogge_Stone_inclusive_scan(float *X, float *Y, int n, float *S){

    __shared__ float XY[SECTION_SIZE];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;

    // Load elements into shared memory
    XY[tx] = (i < n) ? X[i] : 0;

    // Perform scan operation
    for (unsigned int stride = 1; stride < blockDim.x; stride <<= 1){

        __syncthreads();

        if (tx >= stride) XY[tx] += XY[tx - stride];
    }

    // Write results
  
    // Write result to global memory
    if (i < n) Y[i] = XY[tx];

    // Write last element to global memory
    if (S != NULL && tx == blockDim.x - 1)
        S[blockIdx.x] = XY[SECTION_SIZE - 1];

}

//----------------------------------------------------------------------


// Exercise 5
// This kernel can handle up to 2 * SECTION_SIZE elements
__global__
void Brent_Kung_inclusive_scan(float *X, float *Y, int n, float *S){

    __shared__ float XY[2 * SECTION_SIZE];

    int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;

    // Load elements into shared memory
    XY[tx] = (i < n) ? X[i] : 0;
    XY[tx + blockDim.x] = (i + blockDim.x < n) ? X[i + blockDim.x] : 0;

    // Perform scan operation (Phase 1 - Reduction)
    for (unsigned int stride = 1; stride <= blockDim.x; stride <<= 1){

        __syncthreads();

        unsigned int index = (tx + 1) * stride * 2 - 1;
        if(index < 2 * SECTION_SIZE) XY[index] += XY[index - stride];
    }

    // Perform scan operation (Phase 2 - Reverse-Tree)
    for (unsigned int stride = SECTION_SIZE / 2; stride > 0; stride >>= 1){

        __syncthreads();

        unsigned int index = (tx + 1) * stride * 2 - 1;
        if(index + stride < 2 * SECTION_SIZE) XY[index + stride] += XY[index];
    }
    __syncthreads();

    // Write results
  
    // Write result to global memory
    if(i < n) Y[i] = XY[tx];
    if(i + blockDim.x < n) Y[i + blockDim.x] = XY[tx + blockDim.x];

    // Write last element to global memory
    if (S != NULL && tx == blockDim.x - 1)
        S[blockIdx.x] = XY[2 * SECTION_SIZE - 1];
}

//----------------------------------------------------------------------


// Exercise 5
// This kernel can handle up to SHARED_MEM_SIZE elements
__global__
void three_phase_parallel_inclusive_scan(float *X, float *Y, int n, float *S){

    __shared__ float XY[SHARED_MEM_SIZE];

    int start_pos = blockIdx.x * SHARED_MEM_SIZE;
    int tx = threadIdx.x;

    for(int j = tx; j < SHARED_MEM_SIZE; j += blockDim.x) 
        XY[j] = (start_pos + j < n ? X[start_pos + j] : 0);
    __syncthreads();

    // Phase 1
    // Scan each section
    int size = SHARED_MEM_SIZE / blockDim.x;
    int start = size * tx;
    int stop = start + size;
    float acc = 0;
    if(start_pos + start < n) { // Threads that all their values are 0 do not execute (start > n)
        for(int i = start; i < stop; ++i) {
            acc += XY[i];
            XY[i] = acc;
        }
    }
    __syncthreads();

    // Phase 2
    // Use Brent-Kung algorithm to scan the last elements of each section
    for (unsigned int stride = 1; stride <= blockDim.x; stride <<= 1){

        __syncthreads();

        unsigned int index = ((tx + 1) * stride * 2 - 1) * size;
        if(index < SHARED_MEM_SIZE) XY[index] += XY[index - (stride * size)];
    }

    for (unsigned int stride = SHARED_MEM_SIZE / 4; stride > 0; stride >>= 1){

        __syncthreads();

        unsigned int index = (tx + 1) * stride * 2 - 1;
        if(index + stride < SHARED_MEM_SIZE) XY[index + stride] += XY[index];
    }
    __syncthreads();

    // Phase 3
    // Add the last elements of each section to the next section
    if(tx != 0) {
        int value = XY[start - 1];
        for(int i = start; i < stop - 1; ++i) 
            XY[i] += value;
    }
    __syncthreads();

    // Write results
    
    // Write result to global memory
    for(int i = tx; i < SHARED_MEM_SIZE; i += blockDim.x) 
        if(start_pos + i < n) 
            Y[start_pos + i] = XY[i];
    
    // Write last element to global memory
    if (S != NULL && tx == blockDim.x - 1)
        S[blockIdx.x] = XY[SHARED_MEM_SIZE - 1];
}

//----------------------------------------------------------------------


// Exercise 5
// Add intermediate results to the output array
__global__
void add_intermediate_results(float *Y, int n, float *S){

    int i = (blockIdx.x + 1) * blockDim.x + threadIdx.x;

    // Add intermediate results
    if (i < n) Y[i] += S[blockIdx.x];

}

//----------------------------------------------------------------------


// Exercise 5
// This function can handle up to 2 * SECTION_SIZE * (max grid x-dimension size) elements 
void hierarchical_scan(float *X, float *Y, int n){

    float *d_S;

    int blocks = (n + SECTION_SIZE - 1) / SECTION_SIZE;

    // Allocate memory on the device for the intermediate results
    hipMalloc((void **)&d_S, blocks * sizeof(float));

    // Perform scan operation
    Brent_Kung_inclusive_scan<<<blocks, SECTION_SIZE>>>(X, Y, n, d_S);

    // Perform scan operation on the intermediate results
    if(blocks > 2 * SECTION_SIZE) { // Handle up to max grid x-dimension blocks
        hierarchical_scan(d_S, d_S, blocks);
    }

    Brent_Kung_inclusive_scan<<<1, blocks>>>(d_S, d_S, blocks, NULL);

    // Add the intermediate results to the final results
    add_intermediate_results<<<blocks - 1, 2 * SECTION_SIZE>>>(Y, n, d_S);

    // Free memory on the device
    hipFree(d_S);
}

int main(int argc, char *argv[])
{

    float *h_X, *h_Y;
    float *d_X, *d_Y;
    int n;

    if (argc != 2){
        printf("Usage: ./a.out n_elements\n");
        return 1;
    }

    n = atoi(argv[1]);

    // Allocate memory on host
    h_X = (float *) malloc(n * sizeof(float));
    h_Y = (float *) malloc(n * sizeof(float));

    // Allocate memory on device
    hipMalloc((void **) &d_X, n * sizeof(float));
    hipMalloc((void **) &d_Y, n * sizeof(float));

    // Initialize host memory
    srand(time(NULL));
    for (int i = 0; i < n; i++)
        h_X[i] = rand() / (float) RAND_MAX;

#ifdef DEBUG
    // Show input matrices
    printf("X:\n");
    for (int i = 0; i < n; i++)
        printf("%f ", h_X[i]);
    printf("\n");
#endif

    // Copy host memory to device
    hipMemcpy(d_X, h_X, n * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    if(n <= 2 * SECTION_SIZE)      Brent_Kung_inclusive_scan<<<1, SECTION_SIZE>>>(d_X, d_Y, n, NULL);
    else if (n <= SHARED_MEM_SIZE) three_phase_parallel_inclusive_scan<<<1, 512>>>(d_X, d_Y, n, NULL);
    else                           hierarchical_scan(d_X, d_Y, n);

    // Copy device memory to host
    hipMemcpy(h_Y, d_Y, n * sizeof(float), hipMemcpyDeviceToHost);

#ifdef DEBUG
    // Print results
    printf("Y:\n");
    for (int i = 0; i < n; i++)
        printf("%f ", h_Y[i]);
    printf("\n");
#else
    // Print sum
    printf("%f\n", h_Y[n - 1]);
#endif

    // Free memory
    free(h_X);
    free(h_Y);
    hipFree(d_X);
    hipFree(d_Y);

}