/*/
 *
 *  This is a program to test the performance
 *  of two different kernels, add_v1 and add_v2.
 *  
 *  The difference between the two kernels is that
 *  in add_v1, i is the fastest changing thread dimension,
 *  while in add_v2, j is the fastest changing thread dimension.
 * 
 *  Compile with:
 *      nvcc array_add_on_device.cu
 * 
 *  Run with:
 *     ./a.out
 * 
 *  Check the performance of the two kernels with:
 *      nvprof ./a.out  
 * 
/*/


#include <hip/hip_runtime.h>
#include <cstdio>

#define IDX(row, col, LDA) ((row) * (LDA) + (col))

// computes c(i,j) = a(i,j) + b(i,j)
// In this case i is the fastest changing thread dimension
__global__ void add_v1(int *a, int *b, int *c, int N, int M)
{
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < N && j < M)
    {
        int idx = IDX(i, j, M);
        c[idx] = a[idx] + b[idx];
    }
}

// computes c(i,j) = a(i,j) + b(i,j)
// In this case j is the fastest changing thread dimension
__global__ void add_v2(int *a, int *b, int *c, int N, int M)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N && j < M)
    {
        int idx = IDX(i, j, M);
        c[idx] = a[idx] + b[idx];
    }
}

int main(int argc, char *argv[])
{

    int m, n, t_y, t_x;
    int *a, *b, *c;

    if(argc != 5){
        printf("Usage: %s <m> <n> <t_y> <t_x>\n", argv[0]);
        return 1;
    }

    m = atoi(argv[1]);
    n = atoi(argv[2]);
    t_y = atoi(argv[3]);
    t_x = atoi(argv[4]);

    hipMallocManaged(&a, m * n * sizeof(int));
    hipMallocManaged(&b, m * n * sizeof(int));
    hipMallocManaged(&c, m * n * sizeof(int));

    dim3 threads(t_x, t_y);
    dim3 blocks(n / threads.x, m / threads.y);
    add_v1<<<blocks, threads>>>(a, b, c, n, m);
    add_v2<<<blocks, threads>>>(a, b, c, n, m);

    hipFree(a);
    hipFree(b);
    hipFree(c);

    hipDeviceSynchronize();
    return 0;
}
