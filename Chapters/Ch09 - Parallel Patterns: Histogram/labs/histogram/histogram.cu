#include "hip/hip_runtime.h"

#include <stdio.h>

#define NUM_BINS 4096

#define DEBUG

__global__ void histogram_kernel(unsigned int *input, unsigned int *bins,
                                 unsigned int num_elements,
                                 unsigned int num_bins) {

    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Privatized bins
    extern __shared__ unsigned int bins_s[];
    
    // Initialize privatized bins
    for (unsigned int binIdx = threadIdx.x; binIdx < num_bins; binIdx += blockDim.x) {
        bins_s[binIdx] = 0;
    }
    __syncthreads();

    // Histogram
    for (unsigned int i = tid; i < num_elements; i += blockDim.x * gridDim.x) {
        atomicAdd(&bins_s[input[i]], 1);
    }
    __syncthreads();
    
    // Commit to global memory
    for (unsigned int binIdx = threadIdx.x; binIdx < num_bins; binIdx += blockDim.x) {
        atomicAdd(&bins[binIdx], bins_s[binIdx]);
    }
}

int main(int argc, char *argv[]) {
    
    int inputLength;
    unsigned int *hostInput;
    unsigned int *hostBins;
    unsigned int *deviceInput;
    unsigned int *deviceBins;

    if(argc != 2) {
        printf("Usage: %s <inputLength>\n", argv[0]);
        return 0;
    }

    // Read input length
    inputLength = atoi(argv[1]);

    // Allocate host memory
    hostInput = (unsigned int *)malloc(inputLength * sizeof(unsigned int));
    hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(unsigned int));

    // Initialize input
    srand(time(NULL));
    for (int i = 0; i < inputLength; i++) {
        hostInput[i] = rand() % NUM_BINS;
    }

#ifdef DEBUG
    printf("Input:\n");
    for (int i = 0; i < inputLength; i++) {
        printf("%d ", hostInput[i]);
    }
    printf("\n\n");
#endif

    // Allocate device memory
    hipMalloc((void **)&deviceInput, inputLength * sizeof(unsigned int));
    hipMalloc((void **)&deviceBins, NUM_BINS * sizeof(unsigned int));

    // Copy input to device
    hipMemcpy(deviceInput, hostInput, inputLength * sizeof(unsigned int),hipMemcpyHostToDevice);

    // zero out bins
    hipMemset(deviceBins, 0, NUM_BINS * sizeof(unsigned int));

    // Launch histogram kernel on the bins
    dim3 blockDim(256), gridDim(64);
    histogram_kernel<<<gridDim, blockDim, NUM_BINS * sizeof(unsigned int)>>>(
        deviceInput, deviceBins, inputLength, NUM_BINS);

    // Copy bins to host
    hipMemcpy(hostBins, deviceBins, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost);

#ifdef DEBUG
    printf("Bins:\n");
    for (int i = 0; i < NUM_BINS; i++) {
        printf("%d ", hostBins[i]);
    }
    printf("\n");
#endif

    hipFree(deviceInput);
    hipFree(deviceBins);
    
    free(hostBins);
    free(hostInput);

    return 0;
}