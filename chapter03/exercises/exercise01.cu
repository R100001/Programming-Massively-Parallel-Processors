
// A.
#include <stdio.h>
#include <hip/hip_runtime.h>


//#define DEBUG


// B.
__global__
void matrix_addition_B(float *C, float *A, float *B, int n)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i >= n && j >= n) return;

    C[i * n + j] = A[i * n + j] + B[i * n + j];
}


// C.
__global__
void matrix_addition_C(float *C, float *A, float *B, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i >= n) return;

    for (int j = 0; j < n; j++)
    {
        C[i * n + j] = A[i * n + j] + B[i * n + j];
    }
}


// D.
__global__
void matrix_addition_D(float *C, float *A, float *B, int n)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (j >= n) return;

    for (int i = 0; i < n; i++)
    {
        C[i * n + j] = A[i * n + j] + B[i * n + j];
    }
}



int main(int argc, char **argv)
{
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    int n;

    if (argc != 2){
        printf("Usage: ./a.out <n>\n");
        return 1;
    }

    n = atoi(argv[1]);

    // initialize the input matrices
    h_A = (float *)malloc(n * n * sizeof(float));
    h_B = (float *)malloc(n * n * sizeof(float));
    h_C = (float *)malloc(n * n * sizeof(float));

    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            h_A[i * n + j] = 1.0f;
            h_B[i * n + j] = 2.0f;
            h_C[i * n + j] = 0.0f;
        }
    }

    // allocate device memory for the input and output matrices
    hipMalloc((void **)&d_A, n * n * sizeof(float));
    hipMalloc((void **)&d_B, n * n * sizeof(float));
    hipMalloc((void **)&d_C, n * n * sizeof(float));

    // transfer input data to device
    hipMemcpy(d_A, h_A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, n * n * sizeof(float), hipMemcpyHostToDevice);

    // launch the kernel

    // B.
    dim3 dimBlock(32, 16);
    dim3 dimGrid(ceil(n / 32.0f), ceil(n / 16.0f));
    matrix_addition_B<<<dimGrid, dimBlock>>>(d_C, d_A, d_B, n);
    
    // C.
    //dim3 dimBlock(128);
    //dim3 dimGrid(ceil(n / 128.0f));
    //matrix_addition_C<<<dimGrid, dimBlock>>>(d_C, d_A, d_B, n);

    // D.
    //dim3 dimBlock(128);
    //dim3 dimGrid(ceil(n / 128.0f));
    //matrix_addition_D<<<dimGrid, dimBlock>>>(d_C, d_A, d_B, n);

    hipDeviceSynchronize();

    // transfer output data to host
    hipMemcpy(h_C, d_C, n * n * sizeof(float), hipMemcpyDeviceToHost);

#ifdef DEBUG
    // print the result
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            printf("%f ", h_C[i * n + j]);
        }
        printf("\n");
    }
#endif

    // free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}