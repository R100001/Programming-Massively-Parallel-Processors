#include "hip/hip_runtime.h"
/*/
 * 
 *  A program that blurs an image using the mean of the surrounding pixels
 * 
 *  Compile with:
 *      nvcc image_blur.cu
 * 
 *  Run with:
 *     ./a.out <block_size>
 * 
/*/


#include <cstdio>
#include <cstdlib>

#define BLUR_SIZE 1

__global__
void blur_kernel(unsigned char* in, unsigned char* out, int width, int height){

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(col >= width || row >= height) return;

    int pixVal = 0;
    int pixels = 0;

    // Computer the average of the neighboring pixels
    for(int blurrow = -BLUR_SIZE; blurrow < BLUR_SIZE + 1; ++blurrow){
        for(int blurcol = -BLUR_SIZE; blurcol < BLUR_SIZE + 1; ++blurcol){
            int currow = row + blurrow;
            int curcol = col + blurcol;

            // Check if the current pixel is in the image
            if(curcol < 0 || curcol >= width || currow < 0 || currow >= height) continue;

            pixVal += in[currow * width + curcol];
            ++pixels; // Count the number of pixel values that have been added
        }
    }
    // printf("%d %d\n", pixVal, pixels);
    // Write out the result for this pixel
    out[row * width + col] = (unsigned char) ((float)pixVal / pixels);
}


int main(int argc, char* argv[]){

    int block_size;
    int image_width;
    int image_height;
    int size;
    unsigned char* h_input_image, *h_output_image;
    unsigned char* d_input_image, *d_output_image;

    if(argc != 2){
        printf("Usage: %s <block_size>\n", argv[0]);
        return 1;
    }
    block_size = strtol(argv[1], NULL, 10);

    // Set the width and height of the image
    image_width = 20;
    image_height = 10;
    size = image_width * image_height;

    // Allocate memory for the input and output images and initialize the input image
    h_input_image = (unsigned char*) malloc(size * sizeof(unsigned char));
    h_output_image = (unsigned char*) malloc(size * sizeof(unsigned char));
    srand(time(NULL));
    for(int i = 0; i < size; ++i)
        h_input_image[i] = rand() % 256;

    // Show the input image
    for(int i = 0; i < image_height; ++i){
        for(int j = 0; j < image_width; ++j)
            printf("%3d ", h_input_image[i * image_width + j]);
        printf("\n");
    }
    printf("\n");

    // Copy the input image to the device
    hipMalloc((void**) &d_input_image, size * sizeof(unsigned char));
    hipMalloc((void**) &d_output_image, size * sizeof(unsigned char));
    hipMemcpy(d_input_image, h_input_image, size * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Blur the image
    dim3 dimGrid(ceil((float)image_width/block_size), ceil((float)image_height/block_size), 1);
    dim3 dimBlock(block_size, block_size, 1);
    blur_kernel<<<dimGrid, dimBlock>>>(d_input_image, d_output_image, image_width, image_height);

    // Copy the output back to the host
    hipMemcpy(h_output_image, d_output_image, size * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Show the output image
    for(int i = 0; i < image_height; ++i){
        for(int j = 0; j < image_width; ++j)
            printf("%3d ", h_output_image[i * image_width + j]);
        printf("\n");
    }

    // Free the device memory
    hipFree(d_input_image);
    hipFree(d_output_image);

    // Free the host memory
    free(h_input_image);
    free(h_output_image);
}