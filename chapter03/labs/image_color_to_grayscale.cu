#include "hip/hip_runtime.h"
/*/
 * 
 *  A program that converts an input image with 3 channels to a grayscale image with 1 channel
 * 
 *  Compile with:
 *      nvcc image_color_to_grayscale.cu
 * 
 *  Run with:
 *     ./a.out <block_size>
 * 
/*/

#include <cstdio>

#define NUM_CHANNELS 3

__global__
void color_to_grayscale_conversion(unsigned char* in, unsigned char* out, int width, int height){

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < 0 || row >= height || col < 0 || col >= width) return;

    int grey_offset = row * width + col;

    int rgb_offset = grey_offset * NUM_CHANNELS;

    unsigned char r = in[rgb_offset + 0];
    unsigned char g = in[rgb_offset + 1];
    unsigned char b = in[rgb_offset + 2];

    out[grey_offset] = (unsigned char)(0.21f * r + 0.71f * g + 0.07f * b);
}


int main(int argc, char* argv[]){

    int block_dim;
    int image_width;
    int image_height;
    int size;
    unsigned char* h_input_image, *h_output_image;
    unsigned char* d_input_image, *d_output_image;

    if(argc != 2){
        printf("Usage: %s <block_dim>\n", argv[0]);
        return 1;
    }
    block_dim = strtol(argv[1], NULL, 10);

    // Set the width and height of the image
    image_width = 20;
    image_height = 10;
    size = image_width * image_height;

    // Allocate memory for the input and output images and initialize the input image
    h_input_image = (unsigned char*) malloc(NUM_CHANNELS * size * sizeof(unsigned char));
    h_output_image = (unsigned char*) malloc(size * sizeof(unsigned char));
    srand(time(NULL));
    for(int i = 0; i < NUM_CHANNELS * size; ++i)
        h_input_image[i] = rand() % 256;

    // Show the input image
    for(int c = 0; c < NUM_CHANNELS; ++c){
        for(int i = 0; i < image_height; ++i){
            for(int j = 0; j < image_width; ++j){
                printf("%3d ", h_input_image[NUM_CHANNELS * (i * image_width + j) + c]);
            }
            printf("\n");
        }
        printf("\n");
    }
    printf("\n");

    // Copy the input image to the device
    hipMalloc((void**) &d_input_image, NUM_CHANNELS * size * sizeof(unsigned char));
    hipMalloc((void**) &d_output_image, size * sizeof(unsigned char));
    hipMemcpy(d_input_image, h_input_image, NUM_CHANNELS * size * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Blur the image
    dim3 dimGrid(ceil((float)image_width/block_dim), ceil((float)image_height/block_dim), 1);
    dim3 dimBlock(block_dim, block_dim, 1);
    color_to_grayscale_conversion<<<dimGrid, dimBlock>>>(d_input_image, d_output_image, image_width, image_height);

    // Copy the output back to the host
    hipMemcpy(h_output_image, d_output_image, size * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Show the output image
    for(int i = 0; i < image_height; ++i){
        for(int j = 0; j < image_width; ++j)
            printf("%3d ", h_output_image[i * image_width + j]);
        printf("\n");
    }

    // Free the device memory
    hipFree(d_input_image);
    hipFree(d_output_image);

    // Free the host memory
    free(h_input_image);
    free(h_output_image);
}