#include "hip/hip_runtime.h"
/*/
 * 
 *  A program that implements vector addition
 * 
 *  Compile with:
 *      nvcc vector_add.cu
 * 
 *  Run with:
 *     ./a.out <block_size> <input_len>
 * 
/*/

#include <stdio.h>

__global__
void vec_add(int *in1, int *in2, int *out, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n)
        out[i] = in1[i] + in2[i];
}


int main(int argc, char* argv[])
{

    int inputLength;
    int *hostInput1;
    int *hostInput2;
    int *hostOutput;
    int *deviceInput1;
    int *deviceInput2;
    int *deviceOutput;

    int blockSize;

    if (argc != 3)
    {
        printf("Usage: %s <block_size> <input_length>\n", argv[0]);
        return 1;
    }

    blockSize = strtol(argv[1], NULL, 10);
    inputLength = strtol(argv[2], NULL, 10);

    // Allocate host memory for the input and output data
    hostInput1 = (int*)malloc(inputLength * sizeof(int));
    hostInput2 = (int*)malloc(inputLength * sizeof(int));
    hostOutput = (int*)malloc(inputLength * sizeof(int));

    // Initialize the host vectors
    srand(time(NULL));
    for (int i = 0; i < inputLength; i++)
    {
        hostInput1[i] = rand() % 100;
        hostInput2[i] = rand() % 100;
    }

    // Allocate device memory for the input and output data
    hipMalloc((void**)&deviceInput1, inputLength * sizeof(int));
    hipMalloc((void**)&deviceInput2, inputLength * sizeof(int));
    hipMalloc((void**)&deviceOutput, inputLength * sizeof(int));

    // Copy the host input data to the device
    hipMemcpy(deviceInput1, hostInput1, inputLength * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceInput2, hostInput2, inputLength * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel
    vec_add<<<ceil((float)inputLength / blockSize), blockSize>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

    // Copy the device output data to the host
    hipMemcpy(hostOutput, deviceOutput, inputLength * sizeof(int), hipMemcpyDeviceToHost);

    // Print the results
    for (int i = 0; i < inputLength; i++)
        printf("%d + %d = %d\n", hostInput1[i], hostInput2[i], hostOutput[i]);

    // Free the device memory
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);

    // Free the host memory
    free(hostInput1);
    free(hostInput2);
    free(hostOutput);
}