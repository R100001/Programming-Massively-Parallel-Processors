
#include <stdio.h>
#include <hip/hip_runtime.h>

//#define DEBUG


__global__
void mat_vec_mul(float *A, float *B, float *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    A[i] = 0;
    for (int k = 0; k < n; k++)
    {
        A[i] += B[i * n + k] * C[k];
    }
}


int main(int argc, char **argv)
{
    int n;

    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    if (argc != 2){
        printf("Usage: ./a.out <n>\n");
        return 1;
    }

    n = atoi(argv[1]);

    // Allocate memory on host
    h_A = (float *)malloc(n * sizeof(float));
    h_B = (float *)malloc(n * n * sizeof(float));
    h_C = (float *)malloc(n * sizeof(float));

    // Initialize host memory
    for (int i = 0; i < n; i++)
    {
        h_C[i] = 1;
        for (int j = 0; j < n; j++)
        {
            h_B[i * n + j] = 1;
        }
    }

    // Allocate memory on device
    hipMalloc((void **)&d_A, n * sizeof(float));
    hipMalloc((void **)&d_B, n * n * sizeof(float));
    hipMalloc((void **)&d_C, n * sizeof(float));

    // Copy host memory to device memory
    hipMemcpy(d_C, h_C, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, n * n * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    dim3 dimBlock(128);
    dim3 dimGrid(ceil(n / 128.0f));
    mat_vec_mul<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, n);

    // Copy device memory to host memory
    hipMemcpy(h_A, d_A, n * sizeof(float), hipMemcpyDeviceToHost);
    
#ifdef DEBUG
    // Print the result
    for (int i = 0; i < n; i++)
    {
        printf("%f ", h_A[i]);
    }
    printf("\n");
#endif

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}