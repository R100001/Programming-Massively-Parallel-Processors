
#include <thrust/device_vector.h>

#include <nvbench/nvbench.cuh>

#include "array_add_kernels.cuh"

void not_coalesced_bench(nvbench::state &state)
{

    // Get benchmark parameters
    const std::size_t axis_size = state.get_int64("Axis Size");
    const dim3 block_dim = dim3(16, 16);
    const std::size_t grid_size = (axis_size + 15) / 16;
    const dim3 grid_dim = dim3(grid_size, grid_size);

    if(grid_dim.x > (1ULL << 31) - 1 || grid_dim.y > (1 << 16) - 1)
    {
        state.skip("Grid size too large");
        return;
    }

    const std::size_t num_elements = axis_size * axis_size;
    
    // Allocate input data:
    thrust::device_vector<float> input1(num_elements);
    thrust::device_vector<float> input2(num_elements);
    thrust::device_vector<float> output(num_elements);

    // Provide throughput information:
    state.add_element_count(2 * num_elements, "Num Elements");
    state.add_global_memory_reads<float>(2 * num_elements, "Memory Reads");
    state.add_global_memory_writes<float>(num_elements, "Memory Writes");

    state.exec([&input1, &input2, &output, axis_size, grid_dim, block_dim](nvbench::launch &launch) {
        array_add_not_coalesced<float><<<grid_dim, block_dim, 0, launch.get_stream()>>>(
        thrust::raw_pointer_cast(input1.data()),
        thrust::raw_pointer_cast(input2.data()),
        thrust::raw_pointer_cast(output.data()),
        axis_size, axis_size);
    });
}

void coalesced_bench(nvbench::state &state)
{

    // Get benchmark parameters
    const std::size_t axis_size = state.get_int64("Axis Size");
    const dim3 block_dim = dim3(16, 16);
    const std::size_t grid_size = (axis_size + 15) / 16;
    const dim3 grid_dim = dim3(grid_size, grid_size);

    if(grid_dim.x > (1ULL << 31) - 1 || grid_dim.y > (1 << 16) - 1)
    {
        state.skip("Grid size too large");
        return;
    }

    const std::size_t num_elements = axis_size * axis_size;
    
    // Allocate input data:
    thrust::device_vector<float> input1(num_elements);
    thrust::device_vector<float> input2(num_elements);
    thrust::device_vector<float> output(num_elements);

    // Provide throughput information:
    state.add_element_count(2 * num_elements, "Num Elements");
    state.add_global_memory_reads<float>(2 * num_elements, "Memory Reads");
    state.add_global_memory_writes<float>(num_elements, "Memory Writes");

    state.exec([&input1, &input2, &output, axis_size, grid_dim, block_dim](nvbench::launch &launch) {
        array_add_coalesced<float><<<grid_dim, block_dim, 0, launch.get_stream()>>>(
        thrust::raw_pointer_cast(input1.data()),
        thrust::raw_pointer_cast(input2.data()),
        thrust::raw_pointer_cast(output.data()),
        axis_size, axis_size);
    });
}

NVBENCH_BENCH(not_coalesced_bench)
    .set_name("Not Coalesced (Different Array Sizes)")
    .add_int64_power_of_two_axis("Axis Size", nvbench::range(1, 13, 2))
    .set_max_noise(0.2)
    .set_timeout(300)
    ;

NVBENCH_BENCH(coalesced_bench)
    .set_name("Coalesced (Different Array Sizes)")
    .add_int64_power_of_two_axis("Axis Size", nvbench::range(1, 13, 2))
    .set_max_noise(0.2)
    .set_timeout(300)
    ;