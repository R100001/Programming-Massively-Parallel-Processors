#include "hip/hip_runtime.h"

#include <iostream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "mat_vec_mul_kernels.cuh"

int main(int argc, char *argv[])
{
    int rows, cols, t_x;

    if(argc != 4){
        std::cout << "Usage: " << argv[0] << " <rows> <cols> <t_x>" << std::endl;
        return 1;
    }

    rows = atoi(argv[1]);
    cols = atoi(argv[2]);
    t_x = atoi(argv[3]);

    thrust::host_vector<float> h_in_mat(rows * cols);
    thrust::host_vector<float> h_in_vec(cols);

    srand(time(NULL));
    for(int i = 0; i < rows * cols; i++){

        h_in_mat[i] = rand() / (float)RAND_MAX;
        
        if(i < cols)
            h_in_vec[i] = rand() / (float)RAND_MAX;
    }


    thrust::host_vector<float> h_out(rows);

    for(int i = 0; i < rows; ++i)
        for(int j = 0; j < cols; ++j)
            h_out[i] += h_in_mat[i * cols + j] * h_in_vec[j];

    thrust::device_vector<float> d_in_mat = h_in_mat;
    thrust::device_vector<float> d_in_vec = h_in_vec;

    thrust::device_vector<float> d_out(rows);

    dim3 block(t_x);
    dim3 grid((rows + block.x - 1) / block.x);
    mat_vec_mul<<<grid, block>>>(
        thrust::raw_pointer_cast(d_out.data()),
        thrust::raw_pointer_cast(d_in_mat.data()),
        thrust::raw_pointer_cast(d_in_vec.data()),
        rows, cols);

    bool success = true;
    for(int i = 0; i < rows; ++i)
        if(abs(h_out[i] - d_out[i]) >= 0.001){
            std::cout << "Error at " << i << ": " << h_out[i] << " != " << d_out[i] << " (Mat Vec Mul)" << std::endl;
            success = false;
            break;
        }
    if(success)
        std::cout << "Success (Mat Vec Mul)" << std::endl;


    mat_vec_mul_tiles<<<grid, block, block.x * sizeof(float)>>>(
        thrust::raw_pointer_cast(d_out.data()),
        thrust::raw_pointer_cast(d_in_mat.data()),
        thrust::raw_pointer_cast(d_in_vec.data()),
        rows, cols);

    
    success = true;
    for(int i = 0; i < rows; ++i)
        if(abs(h_out[i] - d_out[i]) >= 0.001){
            std::cout << "Error at " << i << ": " << h_out[i] << " != " << d_out[i] << " (Mat Vec Mul Tiles)" << std::endl;
            success = false;
            break;
        }
    if(success)   
        std::cout << "Success (Mat Vec Mul Tiles)" << std::endl;

    return 0;
}