#include "hip/hip_runtime.h"

#include <iostream>
#include <string>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "scan_kernels.cuh"


void check_results(thrust::host_vector<int> &h_res, thrust::host_vector<int> d_res, int n, std::string name){

    bool success = true;
    for(int i = 0; i < n; i++)
        if(abs(h_res[i] - d_res[i]) > 0.001){
            std::cout << "Error at " << i << ": " << h_res[i] << " != " << d_res[i] << std::endl;
            success = false;
            break;
        }
    std::cout << (success ? "Success! " : "Failed.. ") << "( " << name << " )" << std::endl;
}


int main(int argc, char *argv[])
{

    int n;

    if(argc != 2)
    {
        std::cout << "Usage: " << argv[0] << " n" << std::endl;
        return 1;
    }

    n = atoi(argv[1]);

    thrust::host_vector<int> h_vec(n);
    thrust::host_vector<int> h_res(n);
    thrust::device_vector<int> d_vec(n);
    thrust::device_vector<int> d_res(n);

    srand(time(NULL));
    for(int i = 0; i < n; ++i)
        h_vec[i] = 1;

    int acc = 0;
    for(int i = 0; i < n; i++){
        acc += h_vec[i];
        h_res[i] = acc;
    }

    d_vec = h_vec;
    hierarchical_scan_using_three_phase<int>(thrust::raw_pointer_cast(d_vec.data()),
                                             thrust::raw_pointer_cast(d_res.data()),
                                             n);

    check_results(h_res, d_res, n, std::string("Hierarchical Three-Phase"));

    return 0;
}