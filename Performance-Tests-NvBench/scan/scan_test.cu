#include "hip/hip_runtime.h"

#include <iostream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "scan_kernels.cuh"


int main(int argc, char *argv[])
{

    int n1, n2, n3, n4;

    if(argc != 5)
    {
        std::cout << "Usage: " << argv[0] << " n1 n2 n3 n4" << std::endl;
        return 1;
    }

    n1 = atoi(argv[1]);
    n2 = atoi(argv[2]);
    n3 = atoi(argv[3]);
    n4 = atoi(argv[4]);

    thrust::host_vector<int> h_vec1(n1), h_vec2(n2), h_vec3(n3), h_vec4(n4);
    thrust::host_vector<int> h_res1(n1), h_res2(n2), h_res3(n3), h_res4(n4);
    thrust::device_vector<int> d_vec1(n1), d_vec2(n2), d_vec3(n3), d_vec4(n4);
    thrust::device_vector<int> d_res1(n1), d_res2(n2), d_res3(n3), d_res4(n4);

    srand(time(NULL));
    for(int i = 0; i < n1; ++i)
        h_vec1[i] = 1;

    for(int i = 0; i < n2; ++i)
        h_vec2[i] = 1;

    for(int i = 0; i < n3; ++i)
        h_vec3[i] = 1;

    for(int i = 0; i < n4; ++i)
        h_vec4[i] = 1;

    int acc = 0;
    for(int i = 0; i < n1; i++){
        acc += h_vec1[i];
        h_res1[i] = acc;
    }

    acc = 0;
    for(int i = 0; i < n2; i++){
        acc += h_vec2[i];
        h_res2[i] = acc;
    }

    acc = 0;
    for(int i = 0; i < n3; i++){
        acc += h_vec3[i];
        h_res3[i] = acc;
    }

    acc = 0;
    for(int i = 0; i < n4; i++){
        acc += h_vec4[i];
        h_res4[i] = acc;
    }

    d_vec1 = h_vec1;
    Kogge_Stone_inclusive_scan<int><<<1, n1, n1 * sizeof(int)>>>(
        thrust::raw_pointer_cast(d_vec1.data()),
        thrust::raw_pointer_cast(d_res1.data()),
        n1, NULL);

    bool success = true;
    for(int i = 0; i < n1; i++)
        if(abs(h_res1[i] - d_res1[i]) > 0.001){
            std::cout << "Error at " << i << ": " << h_res1[i] << " != " << d_res1[i] << std::endl;
            success = false;
        }
    std::cout << (success ? "Success! (Kogge-Stone)" : "Failed.. (Kogge-Stone)") << std::endl;
    
    d_vec2 = h_vec2;
    Brent_Kung_inclusive_scan<int><<<1, 1024, n2 * sizeof(int)>>>(
        thrust::raw_pointer_cast(d_vec2.data()),
        thrust::raw_pointer_cast(d_res2.data()),
        n2, NULL);
        
    success = true;
    for(int i = 0; i < n2; ++i)
        if(abs(h_res2[i] - d_res2[i]) > 0.001){
            std::cout << "Error at " << i << ": " << h_res2[i] << " != " << d_res2[i] << std::endl;
            success = false;
            break;
        }
    std::cout << (success ? "Success! (Brent-Kung)" : "Failed.. (Brent-Kung)") << std::endl;

    d_vec3 = h_vec3;
    three_phase_parallel_inclusive_scan<int><<<1, 512, n3 * sizeof(int)>>>(
        thrust::raw_pointer_cast(d_vec3.data()),
        thrust::raw_pointer_cast(d_res3.data()),
        n3, n3, NULL);

    success = true;
    for(int i = 0; i < n3; i++)
        if(abs(h_res3[i] - d_res3[i]) > 0.001){
            std::cout << "Error at " << i << ": " << h_res3[i] << " != " << d_res3[i] << std::endl;
            success = false;
            break;
        }
    std::cout << (success ? "Success! (Three-Phase)" : "Failed.. (Three-Phase)") << std::endl;

    d_vec4 = h_vec4;
    hierarchical_scan<int>(thrust::raw_pointer_cast(d_vec4.data()),
                           thrust::raw_pointer_cast(d_res4.data()),
                            n4);

    success = true;
    for(int i = 0; i < n4; i++)
        if(abs(h_res4[i] - d_res4[i]) > 0.001){
            std::cout << "Error at " << i << ": " << h_res4[i] << " != " << d_res4[i] << std::endl;
            success = false;
            break;
        }
    std::cout << (success ? "Success! (Hierarchical)" : "Failed.. (Hierarchical)") << std::endl;
}