#include "hip/hip_runtime.h"

#include <iostream>
#include <string>
#include <numeric>
#include <algorithm>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "scan_kernels.cuh"


void check_results(thrust::host_vector<int> &h_res, thrust::host_vector<int> d_res, int n, std::string name){

    bool success = true;
    for(int i = 0; i < n; i++)
        if(abs(h_res[i] - d_res[i]) > 0.001){
            std::cout << "Error at " << i << ": " << h_res[i] << " != " << d_res[i] << std::endl;
            success = false;
            break;
        }
    std::cout << (success ? "Success! " : "Failed.. ") << "( " << name << " )" << std::endl;
}


int main(int argc, char *argv[])
{

    int n1, n2, n3, n4, n5, n6;

    if(argc != 7)
    {
        std::cout << "Usage: " << argv[0] << " n1 n2 n3 n4 n5 n6" << std::endl;
        return 1;
    }

    n1 = atoi(argv[1]);
    n2 = atoi(argv[2]);
    n3 = atoi(argv[3]);
    n4 = atoi(argv[4]);
    n5 = atoi(argv[5]);
    n6 = atoi(argv[6]);

    thrust::host_vector<int> h_vec1(n1), h_vec2(n2), h_vec3(n3), h_vec4(n4), h_vec5(n5), h_vec6(n6);
    thrust::host_vector<int> h_res1(n1), h_res2(n2), h_res3(n3), h_res4(n4), h_res5(n5), h_res6(n6);
    thrust::device_vector<int> d_vec1(n1), d_vec2(n2), d_vec3(n3), d_vec4(n4), d_vec5(n5), d_vec6(n6);
    thrust::device_vector<int> d_res1(n1), d_res2(n2), d_res3(n3), d_res4(n4), d_res5(n5), d_res6(n6);

    std::fill(h_vec1.begin(), h_vec1.end(), 1);
    std::fill(h_vec2.begin(), h_vec2.end(), 1);
    std::fill(h_vec3.begin(), h_vec3.end(), 1);
    std::fill(h_vec4.begin(), h_vec4.end(), 1);
    std::fill(h_vec5.begin(), h_vec5.end(), 1);
    std::fill(h_vec6.begin(), h_vec6.end(), 1);

    thrust::inclusive_scan(h_vec1.begin(), h_vec1.end(), h_res1.begin());
    thrust::inclusive_scan(h_vec2.begin(), h_vec2.end(), h_res2.begin());
    thrust::inclusive_scan(h_vec3.begin(), h_vec3.end(), h_res3.begin());
    thrust::inclusive_scan(h_vec4.begin(), h_vec4.end(), h_res4.begin());
    thrust::inclusive_scan(h_vec5.begin(), h_vec5.end(), h_res5.begin());
    thrust::inclusive_scan(h_vec6.begin(), h_vec6.end(), h_res6.begin());

    std::cout << "Done with CPU scan" << std::endl;

    d_vec1 = h_vec1;
    Kogge_Stone_inclusive_scan<int><<<1, 1024, 2048 * sizeof(int)>>>(
        thrust::raw_pointer_cast(d_vec1.data()), thrust::raw_pointer_cast(d_res1.data()), n1);
    check_results(h_res1, d_res1, n1, std::string("Kogge Stone"));

    d_vec2 = h_vec2;
    Brent_Kung_inclusive_scan<int><<<1, 1024, 4096 * sizeof(int)>>>(thrust::raw_pointer_cast(d_vec2.data()), thrust::raw_pointer_cast(d_res2.data()), n2);
    check_results(h_res2, d_res2, n2, std::string("Brent Kung"));

    d_vec3 = h_vec3;
    three_phase_parallel_inclusive_scan<int><<<1, 512, 4096 * sizeof(int)>>>(
        thrust::raw_pointer_cast(d_vec3.data()), thrust::raw_pointer_cast(d_res3.data()), n3, 4096);
    check_results(h_res3, d_res3, n3, std::string("Three Phase Parallel"));
/*
    d_vec4 = h_vec4;
    hierarchical_scan_Kogge_Stone<int>(thrust::raw_pointer_cast(d_vec4.data()), thrust::raw_pointer_cast(d_res4.data()), n4);
    check_results(h_res4, d_res4, n4, std::string("Hierarchical Kogge Stone"));

    d_vec5 = h_vec5;
    hierarchical_scan_Brent_Kung<int>(thrust::raw_pointer_cast(d_vec5.data()), thrust::raw_pointer_cast(d_res5.data()), n5);
    check_results(h_res5, d_res5, n5, std::string("Hierarchical Brent Kung"));

    d_vec6 = h_vec6;
    hierarchical_scan_three_phase<int>(thrust::raw_pointer_cast(d_vec6.data()), thrust::raw_pointer_cast(d_res6.data()), n6);
    check_results(h_res6, d_res6, n6, std::string("Hierarchical Three Phase Parallel"));
*/
    return 0;
}