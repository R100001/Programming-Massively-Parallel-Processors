#include "hip/hip_runtime.h"

#include <iostream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define MAX_MASK_WIDTH 121
__constant__ float M[MAX_MASK_WIDTH];

#include "convolution_kernels.cuh"

void check_res(float *h_out, float *d_out, int img_w, std::string kernel_name){
    bool success = true;
    for(int i = 0; i < img_w; ++i){
        if(fabs(h_out[i] - d_out[i]) > 0.001){
            std::cout << "Error at " << i << ": " << h_out[i] << " != " << d_out[i] << std::endl;
            success = false;
            break;
        }
    }
    std::cout << "Test (" << kernel_name << "): " << (success ? "PASSED" : "FAILED") << std::endl;
}

int main(int argc, char *argv[])
{

    int img_h, img_w, img_c, mask_h, mask_w, tile_w, mask_h_radius, mask_w_radius;

    if(argc != 7){
        std::cout << "Usage: " << argv[0] << " <img_h> <img_w> <img_c> <mask_h> <mask_w> <tile_w>" << std::endl;
        return 0;
    }

    img_h = atoi(argv[1]);
    img_w = atoi(argv[2]);
    img_c = atoi(argv[3]);
    mask_h = atoi(argv[4]);
    mask_w = atoi(argv[5]);
    tile_w = atoi(argv[6]);

    mask_h_radius = mask_h / 2;
    mask_w_radius = mask_w / 2;

    thrust::host_vector<float> h_img(img_h * img_w * img_c);
    thrust::host_vector<float> h_mask(mask_h * mask_w);
    thrust::host_vector<float> h_out(img_h * img_w * img_c);
    thrust::host_vector<float> h_dout(img_h * img_w * img_c);

    srand(time(NULL));
    for(int i = 0; i < img_h * img_w * img_c; ++i)
        h_img[i] = (rand() % 256) / 255.0;

    for(int i = 0; i < mask_h * mask_w; ++i)
        h_mask[i] = (rand() % 256) / 255.0 / (mask_h * mask_w / 4.);

    for(int i = 0; i < img_c; ++i)
        for(int j = 0; j < img_h; ++j)
            for(int k = 0; k < img_w; ++k){
                for(int l = 0; l < mask_h; ++l)
                    for(int m = 0; m < mask_w; ++m)
                        if(j + l - mask_h_radius >= 0 && j + l - mask_h_radius < img_h && k + m - mask_w_radius >= 0 && k + m - mask_w_radius < img_w)
                            h_out[j * img_w * img_c + k * img_c + i] += h_img[(j + l - mask_h_radius) * img_w * img_c + (k + m - mask_w_radius) * img_c + i] * h_mask[l * mask_w + m];
                h_out[j * img_w * img_c + k * img_c + i] = clamp(h_out[j * img_w * img_c + k * img_c + i]);
            }

    thrust::device_vector<float> d_img = h_img;
    thrust::device_vector<float> d_mask = h_mask;
    thrust::device_vector<float> d_out(img_h * img_w * img_c);

    hipMemcpyToSymbol(HIP_SYMBOL(M), thrust::raw_pointer_cast(d_mask.data()), mask_h * mask_w * sizeof(float));

    dim3 block_dim(tile_w, tile_w);
    dim3 grid_dim((img_w + block_dim.x - 1) / block_dim.x, (img_h + block_dim.y - 1) / block_dim.y);
    convolution_2D_basic_kernel<<<grid_dim, block_dim>>>(
        thrust::raw_pointer_cast(d_img.data()),
        thrust::raw_pointer_cast(d_mask.data()),
        thrust::raw_pointer_cast(d_out.data()),
        mask_h, mask_w, img_h, img_w, img_c);
    h_dout = d_out;
    check_res(thrust::raw_pointer_cast(h_out.data()),
              thrust::raw_pointer_cast(h_dout.data()),
              img_h * img_w * img_c, "Convolution 2D Basic Kernel");

    convolution_2D_constant_memory_kernel<<<grid_dim, block_dim>>>(
        thrust::raw_pointer_cast(d_img.data()),
        thrust::raw_pointer_cast(d_out.data()),
        mask_h, mask_w, img_h, img_w, img_c);
    h_dout = d_out;
    check_res(thrust::raw_pointer_cast(h_out.data()),
              thrust::raw_pointer_cast(h_dout.data()),
              img_h * img_w * img_c, "Convolution 2D Constant Memory Kernel");

    grid_dim = dim3((img_w + block_dim.x - mask_w) / (block_dim.x - mask_w + 1),
                    (img_h + block_dim.y - mask_h) / (block_dim.y - mask_h + 1));
    convolution_2D_tiled_kernel<<<grid_dim, block_dim, block_dim.y * block_dim.x * sizeof(float)>>>(
        thrust::raw_pointer_cast(d_img.data()),
        thrust::raw_pointer_cast(d_out.data()),
        mask_h, mask_w, img_h, img_w, img_c);
    h_dout = d_out;
    check_res(thrust::raw_pointer_cast(h_out.data()),
              thrust::raw_pointer_cast(h_dout.data()),
              img_h * img_w * img_c, "Convolution 2D Tiled Kernel");
}