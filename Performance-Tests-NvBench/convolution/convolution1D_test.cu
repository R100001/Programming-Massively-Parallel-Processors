#include "hip/hip_runtime.h"

#include <iostream>
#include <string>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define MAX_MASK_WIDTH 25
__constant__ float M[MAX_MASK_WIDTH];

#include "convolution_kernels.cuh"

void check_res(float *h_out, float *d_out, int img_w, std::string kernel_name){
    bool success = true;
    for(int i = 0; i < img_w; ++i){
        if(fabs(h_out[i] - d_out[i]) > 0.001){
            std::cout << "Error at " << i << ": " << h_out[i] << " != " << d_out[i] << std::endl;
            success = false;
        }
    }
    std::cout << "Test (" << kernel_name << "): " << (success ? "PASSED" : "FAILED") << std::endl;
}

int main(int argc, char *argv[])
{

    int img_w, mask_w, tile_w, mask_w_radius;

    if(argc != 4){
        std::cout << "Usage: " << argv[0] << " <img_w> <mask_w> <tile_w>" << std::endl;
        return 0;
    }

    img_w  = atoi(argv[1]);
    mask_w = atoi(argv[2]);
    tile_w = atoi(argv[3]);

    mask_w_radius = mask_w / 2;

    thrust::host_vector<float> h_img(img_w);
    thrust::host_vector<float> h_mask(mask_w);
    thrust::host_vector<float> h_out(img_w);
    thrust::host_vector<float> h_dout(img_w);

    srand(time(NULL));
    for(int i = 0; i < img_w; ++i)
        h_img[i] = (rand() % 256) / 255.0;

    for(int i = 0; i < mask_w; ++i)
        h_mask[i] = (rand() % 256) / 255.0 / (mask_w / 4.);


    for(int i = 0; i < img_w; ++i){
        for(int m = 0; m < mask_w; ++m)
            if(i + m - mask_w_radius >= 0 && i + m - mask_w_radius < img_w)
                h_out[i] += h_img[i + m - mask_w_radius] * h_mask[m];
        h_out[i] = clamp(h_out[i]);
    }

    thrust::device_vector<float> d_img = h_img;
    thrust::device_vector<float> d_mask = h_mask;
    thrust::device_vector<float> d_out(img_w);

    hipMemcpyToSymbol(HIP_SYMBOL(M), thrust::raw_pointer_cast(d_mask.data()), mask_w * sizeof(float));

    int block_dim = tile_w;
    int grid_dim = (img_w + block_dim - 1) / block_dim;
    convolution_1D_basic_kernel<<<grid_dim, block_dim>>>(thrust::raw_pointer_cast(d_img.data()),
                                                         thrust::raw_pointer_cast(d_mask.data()),
                                                         thrust::raw_pointer_cast(d_out.data()),
                                                         mask_w, img_w);
    h_dout = d_out;
    check_res(thrust::raw_pointer_cast(h_out.data()),
              thrust::raw_pointer_cast(h_dout.data()),
              img_w, std::string("Convolution 1D Basic Kernel"));

    convolution_1D_constant_memory_kernel<<<grid_dim, block_dim>>>(thrust::raw_pointer_cast(d_img.data()),
                                                                  thrust::raw_pointer_cast(d_out.data()),
                                                                  mask_w, img_w);
    h_dout = d_out;
    check_res(thrust::raw_pointer_cast(h_out.data()),
              thrust::raw_pointer_cast(h_dout.data()),
              img_w, std::string("Convolution 1D Constant Memory Kernel"));

    int shared_mem_size = (tile_w + mask_w - 1) * sizeof(float);
    convolution_1D_tiled_kernel<<<grid_dim, block_dim, shared_mem_size>>>(
        thrust::raw_pointer_cast(d_img.data()),
        thrust::raw_pointer_cast(d_out.data()),
        mask_w, img_w);
    h_dout = d_out;
    check_res(thrust::raw_pointer_cast(h_out.data()),
              thrust::raw_pointer_cast(h_dout.data()),
              img_w, std::string("Convolution 1D Tiled Kernel"));

    convolution_1D_tiled_caching_kernel<<<grid_dim, block_dim, block_dim * sizeof(float)>>>(
        thrust::raw_pointer_cast(d_img.data()),
        thrust::raw_pointer_cast(d_out.data()),
        mask_w, img_w);
    h_dout = d_out;
    check_res(thrust::raw_pointer_cast(h_out.data()),
              thrust::raw_pointer_cast(h_dout.data()),
              img_w, std::string("Convolution 1D Tiled Caching Kernel"));
}