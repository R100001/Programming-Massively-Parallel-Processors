#include "hip/hip_runtime.h"

#include <iostream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>


#include "mat_mat_mul_kernels.cuh"

int main(int argc, char *argv[])
{

    int rows1, cols1rows2, cols2, t_x;

    if(argc != 5){
        std::cout << "Usage: " << argv[0] << " <rows1> <cols1rows2> <cols2> <t_x>" << std::endl;
        return 1;
    }

    rows1 = atoi(argv[1]);
    cols1rows2 = atoi(argv[2]);
    cols2 = atoi(argv[3]);
    t_x = atoi(argv[4]);

    thrust::host_vector<float> h_in_mat1(rows1 * cols1rows2);
    thrust::host_vector<float> h_in_mat2(cols1rows2 * cols2);
    thrust::host_vector<float> h_out_host(rows1 * cols2);

    srand(time(NULL));
    for(int i = 0; i < rows1 * cols1rows2; i++)
        h_in_mat1[i] = rand() / (float)RAND_MAX;

    for(int i = 0; i < cols1rows2 * cols2; i++)
        h_in_mat2[i] = rand() / (float)RAND_MAX;

    for(int i = 0; i < rows1; ++i)
        for(int j = 0; j < cols2; ++j)
            for(int k = 0; k < cols1rows2; ++k)
                h_out_host[i * cols2 + j] += h_in_mat1[i * cols1rows2 + k] * h_in_mat2[k * cols2 + j];

    thrust::device_vector<float> d_in_mat1 = h_in_mat1;
    thrust::device_vector<float> d_in_mat2 = h_in_mat2;

    thrust::device_vector<float> d_out(rows1 * cols2);

    dim3 blockDim(t_x, t_x);
    dim3 gridDim((cols2 + blockDim.x - 1) / blockDim.x, (rows1 + blockDim.y - 1) / blockDim.y);
    mat_mat_mul<float><<<gridDim, blockDim>>>(
        thrust::raw_pointer_cast(d_in_mat1.data()),
        thrust::raw_pointer_cast(d_in_mat2.data()),
        thrust::raw_pointer_cast(d_out.data()),
        rows1, cols1rows2, cols1rows2, cols2);

    bool success = true;
    for(int i = 0; i < rows1 * cols2; ++i)
        if(abs(h_out_host[i] - d_out[i]) >= 0.001){
            std::cout << "Error at " << i << ": " << h_out_host[i] << " != " << d_out[i] << " (Mat Mat Mul)" << std::endl;
            success = false;
            break;
        }
    if(success)
        std::cout << "Success (Mat Mat Mul)" << std::endl;


    blockDim = dim3(t_x, t_x);
    gridDim = dim3((cols2 + blockDim.x - 1) / blockDim.x, (rows1 + blockDim.y - 1) / blockDim.y);
    mat_mat_mul_tiles<float><<<gridDim, blockDim, 2 * blockDim.x * blockDim.x * sizeof(float)>>>(
        thrust::raw_pointer_cast(d_in_mat1.data()),
        thrust::raw_pointer_cast(d_in_mat2.data()),
        thrust::raw_pointer_cast(d_out.data()),
        rows1, cols1rows2, cols1rows2, cols2);

    success = true;
    for(int i = 0; i < rows1 * cols2; ++i)
        if(abs(h_out_host[i] - d_out[i]) >= 0.001){
            std::cout << "Error at " << i << ": " << h_out_host[i] << " != " << d_out[i] << " (Mat Mat Mul Tiles)" << std::endl;
            success = false;
            break;
        }
    if(success)
        std::cout << "Success (Mat Mat Mul Tiles)" << std::endl;

    
    blockDim = dim3(t_x, t_x);
    gridDim = dim3((cols2 + blockDim.x - 1) / blockDim.x / 2, (rows1 + blockDim.y - 1) / blockDim.y);
    mat_mat_mul_rec_tiles<float><<<gridDim, blockDim, 3 * blockDim.x * blockDim.x * sizeof(float)>>>(
        thrust::raw_pointer_cast(d_in_mat1.data()),
        thrust::raw_pointer_cast(d_in_mat2.data()),
        thrust::raw_pointer_cast(d_out.data()),
        rows1, cols1rows2, cols1rows2, cols2);

    success = true;
    for(int i = 0; i < rows1 * cols2; ++i)
        if(abs(h_out_host[i] - d_out[i]) >= 0.001){
            std::cout << "Error at " << i << ": " << h_out_host[i] << " != " << d_out[i] << " (Mat Mat Mul Rec Tiles)" << std::endl;
            success = false;
            break;
        }
    if(success)
        std::cout << "Success (Mat Mat Mul Rec Tiles)" << std::endl;

    return 0;

}