#include "hip/hip_runtime.h"

#include <iostream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "reduction_kernels.cuh"

int main(int argc, char *argv[])
{

    int n, t_x;

    if(argc != 3){
        printf("Usage: %s <n> <t_x>\n", argv[0]);
        return 1;
    }

    n = atoi(argv[1]);
    t_x = atoi(argv[2]);

    thrust::host_vector<double> h_A(n);

    srand(time(NULL));
    for(int i = 0; i < n; ++i)
        h_A[i] = rand() / (double)RAND_MAX;

    double h_res = thrust::reduce(h_A.begin(), h_A.end(), 0.0f, thrust::plus<double>());
    

    int temp = n;
    thrust::device_vector<double> d_A;
    thrust::device_vector<double> d_B = h_A;
    dim3 threads(t_x);
    dim3 blocks;
    do {
        
        blocks = dim3((temp + threads.x - 1) / threads.x);

        d_A = d_B;
        d_B.resize(blocks.x);

        reduction1<double><<<blocks, threads, threads.x * sizeof(double)>>>(
            thrust::raw_pointer_cast(d_A.data()),
            temp,
            thrust::raw_pointer_cast(d_B.data()));

        temp = blocks.x;

    } while(temp > 1);

    if(abs(h_res - d_B[0]) > 0.0001)
        std::cout << "Error (Reduction 1): " << h_res << " != " << d_B[0] << std::endl;
    else
        std::cout << "Reduction 1: Success!" << std::endl;


    temp = n;
    d_B = h_A;
    do {
        
        blocks = dim3((temp + threads.x - 1) / threads.x);

        d_A = d_B;
        d_B.resize(blocks.x);

        reduction2<double><<<blocks, threads, threads.x * sizeof(double)>>>(
            thrust::raw_pointer_cast(d_A.data()),
            temp,
            thrust::raw_pointer_cast(d_B.data()));

        temp = blocks.x;

    } while(temp > 1);

    if(abs(h_res - d_B[0]) > 0.0001)
        std::cout << "Error (Reduction 2): " << h_res << " != " << d_B[0] << std::endl;
    else
        std::cout << "Reduction 2: Success!" << std::endl;


    temp = n;
    d_B = h_A;
    do {
        
        blocks = dim3((temp + threads.x - 1) / threads.x);

        d_A = d_B;
        d_B.resize(blocks.x);

        reduction3<double><<<blocks, threads, threads.x * sizeof(double)>>>(
            thrust::raw_pointer_cast(d_A.data()),
            temp,
            thrust::raw_pointer_cast(d_B.data()));

        temp = blocks.x;

    } while(temp > 1);

    if(abs(h_res - d_B[0]) > 0.0001)
        std::cout << "Error (Reduction 3): " << h_res << " != " << d_B[0] << std::endl;
    else
        std::cout << "Reduction 3: Success!" << std::endl;


    temp = n;
    d_B = h_A;
    do {
        
        blocks = dim3((temp + threads.x * 2 - 1) / (threads.x * 2));

        d_A = d_B;
        d_B.resize(blocks.x);

        reduction4<double><<<blocks, threads, threads.x * sizeof(double)>>>(
            thrust::raw_pointer_cast(d_A.data()),
            temp,
            thrust::raw_pointer_cast(d_B.data()));

        temp = blocks.x;

    } while(temp > 1);

    if(abs(h_res - d_B[0]) > 0.0001)
        std::cout << "Error (Reduction 4): " << h_res << " != " << d_B[0] << std::endl;
    else
        std::cout << "Reduction 4: Success!" << std::endl;


    temp = n;
    d_B = h_A;
    do {
        
        blocks = dim3((temp + threads.x * 2 - 1) / (threads.x * 2));

        d_A = d_B;
        d_B.resize(blocks.x);

        reduction5<double><<<blocks, threads, threads.x * sizeof(double)>>>(
            thrust::raw_pointer_cast(d_A.data()),
            temp,
            thrust::raw_pointer_cast(d_B.data()));

        temp = blocks.x;

    } while(temp > 1);

    if(abs(h_res - d_B[0]) > 0.0001)
        std::cout << "Error (Reduction 5): " << h_res << " != " << d_B[0] << std::endl;
    else
        std::cout << "Reduction 5: Success!" << std::endl;


    temp = n;
    d_B = h_A;
    do {
        
        blocks = dim3((temp + threads.x * 2 - 1) / (threads.x * 2));

        d_A = d_B;
        d_B.resize(blocks.x);

        reduction6<double><<<blocks, threads, threads.x * sizeof(double)>>>(
            thrust::raw_pointer_cast(d_A.data()),
            temp,
            thrust::raw_pointer_cast(d_B.data()));

        temp = blocks.x;

    } while(temp > 1);

    if(abs(h_res - d_B[0]) > 0.0001)
        std::cout << "Error (Reduction 6): " << h_res << " != " << d_B[0] << std::endl;
    else
        std::cout << "Reduction 6: Success!" << std::endl;


    temp = n;
    d_B = h_A;
    do {
        
        blocks = dim3((temp + threads.x * 2 - 1) / (threads.x * 2));

        d_A = d_B;
        d_B.resize(blocks.x);

        reduction5<double><<<blocks, threads, threads.x * sizeof(double)>>>(
            thrust::raw_pointer_cast(d_A.data()),
            temp,
            thrust::raw_pointer_cast(d_B.data()));

        temp = blocks.x;

    } while(temp > 1);

    if(abs(h_res - d_B[0]) > 0.0001)
        std::cout << "Error (Reduction 7): " << h_res << " != " << d_B[0] << std::endl;
    else
        std::cout << "Reduction 7: Success!" << std::endl;

    return 0;
}