#include "hip/hip_runtime.h"
/*/
 * 
 *  This program implements matrix-matrix multiplication.
 *  The implementation uses tiles to reduce memory usage.
 *  Additionally shared memory is used to reduce global memory accesses.
 *  Global memory accesses are reduced relative to the tile dimensions.
 *  For example using 2x2 tiles, global memory accesses are reduced in half.
 *  For 16x16 tiles, global memory accesses are reduced by a factor of 16.
 *  The tile dimensions are equal to the block dimensions.  
 * 
 *  Compile with:
 *      nvcc basic_matrix_mul.cu
 * 
 *  Run with:
 *     ./a.out <num_rows_A> <num_columns_A> <num_rows_B> <num_columns_B>
 * 
/*/

#include <cstdio>

#define TILE_WIDTH 16

// Compute C = A * B
__global__ void matrix_mul(float *A, float *B, float *C,
                           int num_rows_A, int num_cols_A,
                           int num_rows_B, int num_cols_B,
                           int num_rows_C, int num_cols_C) {
    
    // Initialize shared memory
    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];
    
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    float Pvalue = 0;
    // Loop over the M and N tiles required to compute the P element
    for (int ph = 0; ph < ceil(num_cols_A/(float)TILE_WIDTH); ph++) {

        // Collaborative loading of M and N tiles into shared memory
        if (row < num_rows_A && ph * TILE_WIDTH + tx < num_cols_A)
            ds_M[ty][tx] = A[row * num_cols_A + ph * TILE_WIDTH + tx];
        else
            ds_M[ty][tx] = 0;

        if (col < num_cols_B && ph * TILE_WIDTH + ty < num_rows_B)
            ds_N[ty][tx] = B[(ph * TILE_WIDTH + ty) * num_cols_B + col];
        else
            ds_N[ty][tx] = 0;

        // Synchronize to make sure the tiles are loaded
        __syncthreads();

        // Compute the P element
        for (int k = 0; k < TILE_WIDTH; k++)
            Pvalue += ds_M[ty][k] * ds_N[k][tx];
        
        // Synchronize to make sure the P element is computed
        // before other threads load new tiles
        __syncthreads();
    }

    // Store the P element in C
    if (row < num_rows_C && col < num_cols_C)
        C[row * num_cols_C + col] = Pvalue;
}


int main(int argc, char **argv) {

    float *hostA;
    float *hostB;
    float *hostC;
    float *deviceA;
    float *deviceB;
    float *deviceC;
    int numARows;
    int numAColumns;
    int numBRows;
    int numBColumns;
    int numCRows;
    int numCColumns;

    if (argc != 5){
        printf("Usage: ./a.out <num_rows_A> <num_columns_A> <num_rows_B> <num_columns_B>\n");
        return 1;
    }

    numARows = atoi(argv[1]);
    numAColumns = atoi(argv[2]);
    numBRows = atoi(argv[3]);
    numBColumns = atoi(argv[4]);
    
    if(numAColumns != numBRows) {
        printf("Number of columns in A must be the same as the number of rows in B\n");
        return 1;
    }

    numCRows = numARows;
    numCColumns = numBColumns;

    // Allocate memory on host
    hostA = (float *) malloc(numARows * numAColumns * sizeof(float));
    hostB = (float *) malloc(numBRows * numBColumns * sizeof(float));
    hostC = (float *) malloc(numCRows * numCColumns * sizeof(float));

    // Allocate memory on device
    hipMalloc((void **) &deviceA, numARows * numAColumns * sizeof(float));
    hipMalloc((void **) &deviceB, numBRows * numBColumns * sizeof(float));
    hipMalloc((void **) &deviceC, numCRows * numCColumns * sizeof(float));

    // Initialize host memory
    srand(time(NULL));
    for (int i = 0; i < numARows; i++)
        for (int j = 0; j < numAColumns; j++)
            hostA[i * numAColumns + j] = rand() / (float) RAND_MAX;

    for (int i = 0; i < numBRows; i++)
        for (int j = 0; j < numBColumns; j++)
            hostB[i * numBColumns + j] = rand() / (float) RAND_MAX;

    // Show input matrices
    printf("A:\n");
    for (int i = 0; i < numARows; i++) {
        for (int j = 0; j < numAColumns; j++)
            printf("%f ", hostA[i * numAColumns + j]);
        printf("\n");
    }
    printf("\n");

    printf("B:\n");
    for (int i = 0; i < numBRows; i++) {
        for (int j = 0; j < numBColumns; j++)
            printf("%f ", hostB[i * numBColumns + j]);
        printf("\n");
    }
    printf("\n");

    // Copy host memory to device
    hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
    dim3 gridDim(ceil((float)numCColumns / blockDim.x), ceil((float)numCRows / blockDim.y));
    matrix_mul<<<gridDim, blockDim>>>(deviceA, deviceB, deviceC,
                                      numARows, numAColumns,
                                      numBRows, numBColumns,
                                      numCRows, numCColumns);

    // Copy device memory to host
    hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float), hipMemcpyDeviceToHost);

    // Print results
    printf("C:\n");
    for (int i = 0; i < numCRows; i++) {
        for (int j = 0; j < numCColumns; j++)
            printf("%f ", hostC[i * numCColumns + j]);
        printf("\n");
    }

    // Free memory
    free(hostA);
    free(hostB);
    free(hostC);
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    return 0;
}