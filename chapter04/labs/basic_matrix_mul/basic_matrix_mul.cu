#include "hip/hip_runtime.h"
/*/
 * 
 *  This program implements matrix-matrix multiplication in its simplest form.  
 * 
 *  Compile with:
 *      nvcc basic_matrix_mul.cu
 * 
 *  Run with:
 *     ./a.out <num_rows_A> <num_columns_A> <num_rows_B> <num_columns_B>
 * 
/*/

#include <cstdio>

// Compute A * B
// A and B can have arbitrary dimensions
// Sgemm stands for single precision general matrix-matrix multiply
__global__ void sgemm(float *A, float *B, float *C, int numARows, int numAColumns, 
                                                    int numBRows, int numBColumns) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= numARows || col >= numBColumns) return;

    float sum = 0;
    for (int k = 0; k < numAColumns; k++)
        sum += A[row * numAColumns + k] * B[k * numBColumns + col];

    C[row * numBColumns + col] = sum;
}


int main(int argc, char **argv) {

    float *hostA;
    float *hostB;
    float *hostC;
    float *deviceA;
    float *deviceB;
    float *deviceC;
    int numARows;
    int numAColumns;
    int numBRows;
    int numBColumns;
    int numCRows;
    int numCColumns;

    if (argc != 5){
        printf("Usage: ./a.out <num_rows_A> <num_columns_A> <num_rows_B> <num_columns_B>\n");
        return 1;
    }

    numARows = atoi(argv[1]);
    numAColumns = atoi(argv[2]);
    numBRows = atoi(argv[3]);
    numBColumns = atoi(argv[4]);
    
    if(numAColumns != numBRows) {
        printf("Number of columns in A must be the same as the number of rows in B\n");
        return 1;
    }

    numCRows = numARows;
    numCColumns = numBColumns;

    // Allocate memory on host
    hostA = (float *) malloc(numARows * numAColumns * sizeof(float));
    hostB = (float *) malloc(numBRows * numBColumns * sizeof(float));
    hostC = (float *) malloc(numCRows * numCColumns * sizeof(float));

    // Allocate memory on device
    hipMalloc((void **) &deviceA, numARows * numAColumns * sizeof(float));
    hipMalloc((void **) &deviceB, numBRows * numBColumns * sizeof(float));
    hipMalloc((void **) &deviceC, numCRows * numCColumns * sizeof(float));

    // Initialize host memory
    srand(time(NULL));
    for (int i = 0; i < numARows; i++)
        for (int j = 0; j < numAColumns; j++)
            hostA[i * numAColumns + j] = rand() / (float) RAND_MAX;

    for (int i = 0; i < numBRows; i++)
        for (int j = 0; j < numBColumns; j++)
            hostB[i * numBColumns + j] = rand() / (float) RAND_MAX;

    // Show input matrices
    printf("A:\n");
    for (int i = 0; i < numARows; i++) {
        for (int j = 0; j < numAColumns; j++)
            printf("%f ", hostA[i * numAColumns + j]);
        printf("\n");
    }
    printf("\n");

    printf("B:\n");
    for (int i = 0; i < numBRows; i++) {
        for (int j = 0; j < numBColumns; j++)
            printf("%f ", hostB[i * numBColumns + j]);
        printf("\n");
    }
    printf("\n");

    // Copy host memory to device
    hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 blockDim(16, 16);
    dim3 gridDim(ceil((float)numCColumns / blockDim.x), ceil((float)numCRows / blockDim.y));
    sgemm<<<gridDim, blockDim>>>(deviceA, deviceB, deviceC,
                                 numARows, numAColumns,
                                 numBRows, numBColumns);

    // Copy device memory to host
    hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float), hipMemcpyDeviceToHost);

    // Print results
    printf("C:\n");
    for (int i = 0; i < numCRows; i++) {
        for (int j = 0; j < numCColumns; j++)
            printf("%f ", hostC[i * numCColumns + j]);
        printf("\n");
    }

    // Free memory
    free(hostA);
    free(hostB);
    free(hostC);
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    return 0;
}