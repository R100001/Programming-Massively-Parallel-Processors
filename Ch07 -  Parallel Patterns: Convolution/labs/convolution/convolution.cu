#include "hip/hip_runtime.h"

#include <stdio.h>

#define DEBUG


#define Mask_width 5
#define Mask_radius Mask_width / 2
#define TILE_WIDTH 16
#define w (TILE_WIDTH + Mask_width - 1)
#define clamp(x) (min(max((x), 0.0), 1.0))

__global__ void convolution(float *I, const float *__restrict__ M, float *P, 
                            int channels, int width, int height) {

    __shared__ float N_ds[w][w];
    
    int k;
    for (k = 0; k < channels; k++) {
        
        // First batch loading
        int dest = threadIdx.y * TILE_WIDTH + threadIdx.x;
        int destY = dest / w;
        int destX = dest % w;
        int srcY = blockIdx.y * TILE_WIDTH + destY - Mask_radius;
        int srcX = blockIdx.x * TILE_WIDTH + destX - Mask_radius;
        int src = (srcY * width + srcX) * channels + k;
        
        if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width) {
            N_ds[destY][destX] = I[src];
        } else {
            N_ds[destY][destX] = 0;
        }
        
        // Second batch loading
        dest = threadIdx.y * TILE_WIDTH + threadIdx.x + TILE_WIDTH * TILE_WIDTH;
        destY = dest / w;
        destX = dest % w;
        srcY = blockIdx.y * TILE_WIDTH + destY - Mask_radius;
        srcX = blockIdx.x * TILE_WIDTH + destX - Mask_radius;
        src = (srcY * width + srcX) * channels + k;
        
        if (destY < w) {

            if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width) {
                N_ds[destY][destX] = I[src];
            } 
            else {
                N_ds[destY][destX] = 0;
            }
        }
        __syncthreads();
    
        float accum = 0;
        int y, x;
        for (y = 0; y < Mask_width; y++) {
            for (x = 0; x < Mask_width; x++) {
                accum += N_ds[threadIdx.y + y][threadIdx.x + x] * M[y * Mask_width + x];
            }
        }
        
        y = blockIdx.y * TILE_WIDTH + threadIdx.y;
        x = blockIdx.x * TILE_WIDTH + threadIdx.x;
        if (y < height && x < width)
            P[(y * width + x) * channels + k] = clamp(accum);
        __syncthreads();
    }
}


int main(int argc, char *argv[]) {

    int maskRows;
    int maskColumns;
    int imageChannels;
    int imageWidth;
    int imageHeight;
    float *hostInputImageData;
    float *hostOutputImageData;
    float *hostMaskData;
    float *deviceInputImageData;
    float *deviceOutputImageData;
    float *deviceMaskData;

    if (argc != 4){
        printf("Usage: %s <image_height> <image_width> <image_channels>\n", argv[0]);
        return 0;
    }

    imageHeight = atoi(argv[1]);
    imageWidth = atoi(argv[2]);
    imageChannels = atoi(argv[3]);
    maskRows = Mask_width;
    maskColumns = Mask_width;

    // Allocate host memory
    hostInputImageData = (float *)malloc(imageHeight * imageWidth * imageChannels * sizeof(float));
    hostOutputImageData = (float *)malloc(imageHeight * imageWidth * imageChannels * sizeof(float));
    hostMaskData = (float *)malloc(maskRows * maskColumns * sizeof(float));

    // Initialize the input and mask
    srand(time(NULL));
    for (int i = 0; i < imageHeight; ++i)
        for (int j = 0; j < imageWidth; ++j)
            for(int k = 0; k < imageChannels; ++k)
                hostInputImageData[(i * imageWidth + j) * imageChannels + k] = (float)rand() / (float)RAND_MAX;

    for (int i = 0; i < maskRows; ++i)
        for (int j = 0; j < maskColumns; ++j)
            hostMaskData[i * maskColumns + j] = ((float)(rand() % 256) / 255.0f) / (Mask_width * Mask_width / 4.0f); 

#ifdef DEBUG
    printf("Input Image:\n");

    for(int k = 0; k < imageChannels; ++k) {
        for (int i = 0; i < imageHeight; ++i) {
            for (int j = 0; j < imageWidth; ++j)
                printf("%f ", hostInputImageData[(i * imageWidth + j) * imageChannels + k]);
            printf("\n");
        }
        printf("\n\n");
    }
    printf("\n\n");

    printf("Mask:\n");
    for (int i = 0; i < maskRows; ++i) {
        for (int j = 0; j < maskColumns; ++j)
            printf("%f ", hostMaskData[i * maskColumns + j]);
        printf("\n");
    }
    printf("\n\n");
#endif

    // Allocate device memory
    hipMalloc((void **)&deviceInputImageData, imageHeight * imageWidth * imageChannels * sizeof(float));
    hipMalloc((void **)&deviceOutputImageData, imageHeight * imageWidth * imageChannels * sizeof(float));
    hipMalloc((void **)&deviceMaskData, maskRows * maskColumns * sizeof(float));

    // Copy host memory to device
    hipMemcpy(deviceInputImageData, hostInputImageData, imageHeight * imageWidth * imageChannels * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceMaskData, hostMaskData, maskRows * maskColumns * sizeof(float), hipMemcpyHostToDevice);

    
    // Launch kernel
    dim3 dimGrid((float)(imageWidth + TILE_WIDTH - 1) / TILE_WIDTH,
                 (float)(imageHeight + TILE_WIDTH - 1) / TILE_WIDTH);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    convolution<<<dimGrid, dimBlock>>>(deviceInputImageData, deviceMaskData,
                                       deviceOutputImageData, imageChannels,
                                       imageWidth, imageHeight);

    // Copy result from device to host
    hipMemcpy(hostOutputImageData, deviceOutputImageData, 
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyDeviceToHost);

#ifdef DEBUG
    printf("Output Image:\n");
    for(int k = 0; k < imageChannels; ++k) {
        for (int i = 0; i < imageHeight; ++i) {
            for (int j = 0; j < imageWidth; ++j)
                printf("%f ", hostOutputImageData[(i * imageWidth + j) * imageChannels + k]);
            printf("\n");
        }
        printf("\n\n");
    }
    printf("\n\n");
#endif

    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    hipFree(deviceMaskData);
    
    free(hostMaskData);

    return 0;
}