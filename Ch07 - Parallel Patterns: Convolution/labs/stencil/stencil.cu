#include "hip/hip_runtime.h"

#include <stdio.h>

#define DEBUG

#define dx 32
#define dy 32
#define TILE_SIZE 32

#define out(i, j, k) _out[((i)*width + (j)) * depth + (k)]
#define in(i, j, k) _in[((i)*width + (j)) * depth + (k)]

__host__ __device__ float Clamp(float val, float start, float end) {
    return max(min(val, end), start);
}

void stencil_cpu(float *_out, float *_in, int width, int height, int depth) {

    float res;
    for (int i = 1; i < height - 1; ++i) {
        for (int j = 1; j < width - 1; ++j) {
            for (int k = 1; k < depth - 1; ++k) {
                res = in(i, j, k + 1) + in(i, j, k - 1) + in(i, j + 1, k) +
                      in(i, j - 1, k) + in(i + 1, j, k) + in(i - 1, j, k) -
                      6 * in(i, j, k);
                out(i, j, k) = Clamp(res, 0, 255);
            }
        }
    }
}

__global__ void stencil(float *_out, float *_in, int width, int height,
                        int depth) {

    int k = blockIdx.z * TILE_SIZE + threadIdx.z;
    int j = blockIdx.x * TILE_SIZE + threadIdx.x;
    
    __shared__ float ds_A[TILE_SIZE][TILE_SIZE];
    
    float bottom = in(0, j, k);
    float current = in(1, j, k);
    float top = in(2, j, k);
    
    ds_A[threadIdx.z][threadIdx.x] = current;
    __syncthreads();
    
    for (int i = 1; i < height - 1; i++) 
    {
        float temp = 0;
        if (k < depth - 1 && k > 0 && j < width - 1 && j > 0) {

            temp = bottom + top;
            if (threadIdx.z > 0) {
                temp += ds_A[threadIdx.z - 1][threadIdx.x];
            }
            else {
                temp += in(i, j, k - 1);
            }

            if (threadIdx.z < TILE_SIZE - 1) {
                temp += ds_A[threadIdx.z + 1][threadIdx.x];
            }
            else {
                temp += in(i, j, k + 1);
            }

            if (threadIdx.x > 0) {
                temp += ds_A[threadIdx.z][threadIdx.x - 1];
            }
            else {
                temp += in(i, j - 1, k);
            }

            if (threadIdx.x < TILE_SIZE - 1) {
                temp += ds_A[threadIdx.z][threadIdx.x + 1];
            }
            else {
                temp += in(i, j + 1, k);
            }
            temp -= 6 * current;
            printf("%f ", temp);
            out(i, j, k) = Clamp(temp, 0, 255);
        }

        bottom = current;
        __syncthreads();
        
        ds_A[threadIdx.z][threadIdx.x] = top;
        __syncthreads();
        
        current = top;
        top = in(i + 2, j, k);
    }
}

static void launch_stencil(float *deviceOutputData, float *deviceInputData,
                           int width, int height, int depth)
{
    const unsigned int zBlocks = (depth - 1) / TILE_SIZE + 1;
    const unsigned int xBlocks = (width - 1) / TILE_SIZE + 1;
    dim3 GridD(xBlocks, 1, zBlocks);
    dim3 BlockD(TILE_SIZE, 1, TILE_SIZE);
    stencil<<<GridD, BlockD>>>(deviceOutputData, deviceInputData, width,
                               height, depth);
}

int main(int argc, char *argv[])
{

    int width;
    int height;
    int depth;
    float *hostInputData;
    float *hostOutputData;
    float *deviceInputData;
    float *deviceOutputData;

    if (argc != 4) {
        printf("Usage: %s <width> <height> <depth>\n", argv[0]);
        return 0;
    }

    width = atoi(argv[1]);
    height = atoi(argv[2]);
    depth = atoi(argv[3]);

    // Allocate host memory
    hostInputData = (float *)malloc(width * height * depth * sizeof(float));
    hostOutputData = (float *)malloc(width * height * depth * sizeof(float));

    // Initialize host memory
    srand(time(NULL));
    for (int i = 0; i < width * height * depth; i++) {
        hostInputData[i] = (rand() / (float)RAND_MAX) * 255;
    }

#ifdef DEBUG
    printf("Input:\n");
    for (int i = 0; i < width * height * depth; i++) {
        printf("%.3f ", hostInputData[i]);
    }
    printf("\n\n");
#endif

    // Allocate device memory
    hipMalloc((void **)&deviceInputData, width * height * depth * sizeof(float));
    hipMalloc((void **)&deviceOutputData, width * height * depth * sizeof(float));

    // Copy data from host to device
    hipMemcpy(deviceInputData, hostInputData,
               width * height * depth * sizeof(float),
               hipMemcpyHostToDevice);

    // Launch kernel
    launch_stencil(deviceOutputData, deviceInputData, width, height, depth);

    // Copy data from device to host
    hipMemcpy(hostOutputData, deviceOutputData,
               width * height * depth * sizeof(float),
               hipMemcpyDeviceToHost);

#ifdef DEBUG
    printf("Output:\n");
    for (int i = 0; i < width * height * depth; i++) {
        printf("%.3f ", hostOutputData[i]);
    }
    printf("\n");
#endif

    hipFree(deviceInputData);
    hipFree(deviceOutputData);

    return 0;
}