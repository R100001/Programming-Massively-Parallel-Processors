#include "hip/hip_runtime.h"

#include <iostream>

#define TILE_DIM 32             // For transpose
#define BLOCK_ROWS TILE_DIM / 4 // For transpose

#define BLOCKSIZE 256
#define MOD 10

#define DEBUG

//-------------------------------------------------------------------------------------------------


void ELL_COO_SpMV(int *h_A, int *h_X, int *h_Y,
                  int rows, int cols);

void JDS_ELL_SpMV(int *h_A, int *h_X, int *h_Y,
                  int rows, int cols);

int count_nonzeros(int *h_A, int *zeros_per_row, int rows, int cols);

void generate_formats_ELL_COO(int *A, int rows, int cols, int max_nonzeros,
                              int *ELL_data, int *ELL_index,
                              int *COO_data, int *COO_col_index, int *COO_row_index);

int split_A(int *splits, int* nonzeros_counts, int rows);


__global__ 
void transpose(const int *idata, int *odata, int rows, int cols);

__global__
void parallel_ELL(int *d_ELL_data_transposed, int *d_ELL_index_transposed, 
                  int *d_X, int* d_Y, 
                  int rows, int max_nonzeros);

__global__
void parallel_count_sort_JDS(int *d_nonzeros_counts, int *d_nonzeros_counts_reordered,
                             int* d_A_reordered, int *d_JDS_index_reordered,
                             int rows, int cols);

__global__
void parallel_copy(int *d_JDS_index, int *d_A, int *d_A_reordered,
                   int rows, int cols);

//-------------------------------------------------------------------------------------------------


int main(int argc, char *argv[])
{

    int *h_A, *h_X, *h_Y;
    int rows, cols;
    float zero_prob;
    int algorithm;
    
    if (argc != 5){
        printf("Usage: %s <rows> <columns> <zero prob> <algorithm>\n", argv[0]);
        return 1;
    }

    rows = atoi(argv[1]);
    cols = atoi(argv[2]);
    zero_prob = atof(argv[3]);
    algorithm = atoi(argv[4]);

    if(rows <= 0 || cols <= 0 || zero_prob < 0 || zero_prob > 1){
        printf("Invalid arguments");
        return 1;
    }

    // Allocate memory for the matrix and vector
    h_A = (int*)malloc(rows * cols * sizeof(int));
    h_X = (int*)malloc(cols * sizeof(int));
    h_Y = (int*)malloc(rows * sizeof(int));

    // Fill the matrix with random values
    srand(time(NULL));
    for(int i = 0; i < rows; i++)
        for(int j = 0; j < cols; j++)
            if((float)rand() / RAND_MAX < zero_prob)
                h_A[i * cols + j] = 0;
            else
                h_A[i * cols + j] = rand() % MOD;
    
    // Fill the vector with random values
    for(int i = 0; i < cols; i++)
        h_X[i] = rand() % MOD;
    
#ifdef DEBUG
    // Print the matrix
    printf("A:\n");
    for(int i = 0; i < rows; i++){
        for(int j = 0; j < cols; j++)
            printf("%d ", h_A[i * cols + j]);
        printf("\n");
    }
    printf("\n");

    // Print the vector
    printf("X:\n");
    for(int i = 0; i < cols; i++)
        printf("%d ", h_X[i]);
    printf("\n\n");
#endif

    // Compute the product on the device using ELL-COO format
    if(algorithm == 1) ELL_COO_SpMV(h_A, h_X, h_Y, rows, cols);

    // Compute the product on the device using JDS-ELL format
    if(algorithm == 2) JDS_ELL_SpMV(h_A, h_X, h_Y, rows, cols);

#ifdef DEBUG
    // Print the result
    printf("Y:\n");
    for(int i = 0; i < rows; i++)
        printf("%d ", h_Y[i]);
    printf("\n");
#endif

    // Free host memory
    free(h_A);
    free(h_X);
    free(h_Y);

    return 0;
}


//-------------------------------------------------------------------------------------------------

void ELL_COO_SpMV(int *h_A, int *h_X, int *h_Y,
                  int rows, int cols){

    int *nonzeros_per_row = NULL;
    int total_nonzeros = 0;
    int max_nonzeros = 0;
    int nonzeros_COO = 0;

    int *h_ELL_data = NULL,            *h_ELL_index = NULL;
    int *d_ELL_data = NULL,            *d_ELL_index = NULL;
    int *d_ELL_data_transposed = NULL, *d_ELL_index_transposed = NULL;

    int *h_COO_data =      NULL;
    int *h_COO_col_index = NULL;
    int *h_COO_row_index = NULL;

    int *d_X = NULL;
    int *d_Y = NULL;

    nonzeros_per_row = (int *)malloc(rows * sizeof(int));

    total_nonzeros = count_nonzeros(h_A, nonzeros_per_row, rows, cols);
    max_nonzeros = total_nonzeros / rows;

    for(int i = 0; i < rows; ++i)
        nonzeros_COO += max(nonzeros_per_row[i] - max_nonzeros, 0);

    h_ELL_data =  (int *)malloc(rows * max_nonzeros * sizeof(int));
    h_ELL_index = (int *)malloc(rows * max_nonzeros * sizeof(int));

    memset(h_ELL_data,  0, rows * max_nonzeros * sizeof(int));
    memset(h_ELL_index, 0, rows * max_nonzeros * sizeof(int));
    
    h_COO_data =      (int *)malloc(nonzeros_COO * sizeof(int));
    h_COO_row_index = (int *)malloc(nonzeros_COO * sizeof(int));
    h_COO_col_index = (int *)malloc(nonzeros_COO * sizeof(int));

    memset(h_COO_data,      0, nonzeros_COO * sizeof(int));
    memset(h_COO_row_index, 0, nonzeros_COO * sizeof(int));
    memset(h_COO_col_index, 0, nonzeros_COO * sizeof(int));
    
    generate_formats_ELL_COO(h_A, rows, cols, max_nonzeros,
                             h_ELL_data, h_ELL_index,
                             h_COO_data, h_COO_col_index, h_COO_row_index);

    // Allocate device memory
    hipMalloc((void**)&d_ELL_data,             rows * max_nonzeros * sizeof(int));
    hipMalloc((void**)&d_ELL_data_transposed,  rows * max_nonzeros * sizeof(int));
    hipMalloc((void**)&d_ELL_index,            rows * max_nonzeros * sizeof(int));
    hipMalloc((void**)&d_ELL_index_transposed, rows * max_nonzeros * sizeof(int));

    hipMemcpy(d_ELL_data,  h_ELL_data,  rows * max_nonzeros * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_ELL_index, h_ELL_index, rows * max_nonzeros * sizeof(int), hipMemcpyHostToDevice);

    dim3 dimBlock(TILE_DIM, BLOCK_ROWS);
    dim3 dimGrid((rows + TILE_DIM - 1) / TILE_DIM, (cols + TILE_DIM - 1) / TILE_DIM);
    transpose<<<dimGrid, dimBlock>>>(d_ELL_data,  d_ELL_data_transposed,  rows, max_nonzeros);
    transpose<<<dimGrid, dimBlock>>>(d_ELL_index, d_ELL_index_transposed, rows, max_nonzeros);

    int *h_ELL_data_transposed =  (int *)malloc(rows * max_nonzeros * sizeof(int));
    int *h_ELL_index_transposed = (int *)malloc(rows * max_nonzeros * sizeof(int));

    hipMemcpy(h_ELL_data_transposed,  d_ELL_data_transposed,  rows * max_nonzeros * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_ELL_index_transposed, d_ELL_index_transposed, rows * max_nonzeros * sizeof(int), hipMemcpyDeviceToHost);

    hipMalloc((void **)&d_X, cols * sizeof(int));
    hipMalloc((void **)&d_Y, rows * sizeof(int));
    hipMemcpy(d_X, h_X, cols * sizeof(int), hipMemcpyHostToDevice);

    dimBlock = dim3(BLOCKSIZE);
    dimGrid = dim3((max_nonzeros + dimBlock.x - 1) / dimBlock.x);
    parallel_ELL<<<dimGrid, dimBlock>>>(d_ELL_data_transposed, d_ELL_index_transposed, d_X, d_Y, rows, max_nonzeros);

    // Copy results back to host
    hipMemcpy(h_Y, d_Y, rows * sizeof(int), hipMemcpyDeviceToHost);

    // Perform sequential COO
    for(int i = 0; i < nonzeros_COO; ++i)
        h_Y[h_COO_row_index[i]] += h_COO_data[i] * h_X[h_COO_col_index[i]];

    // Free device memory
    hipFree(d_ELL_data);
    hipFree(d_ELL_index);
    hipFree(d_ELL_data_transposed);
    hipFree(d_ELL_index_transposed);
    hipFree(d_X);
    hipFree(d_Y);

    // Free host memory
    free(nonzeros_per_row);
    free(h_ELL_data);
    free(h_ELL_index);
    free(h_COO_data);
    free(h_COO_col_index);
    free(h_COO_row_index);
}

//-------------------------------------------------------------------------------------------------


void JDS_ELL_SpMV(int *h_A, int *h_X, int *h_Y,
                  int rows, int cols){

    int *h_nonzeros_counts = NULL;
    int *d_nonzeros_counts = NULL;
    int *h_nonzeros_counts_reordered = NULL;
    int *d_nonzeros_counts_reordered = NULL;

    int *d_A_new_order = NULL;
    int *h_JDS_index_reordered = NULL;
    int *d_JDS_index_reordered = NULL;


    int *h_A_reordered = NULL;
    int *d_A = NULL;
    int *d_A_reordered = NULL;
    int *d_X = NULL;

    int *splits = NULL;
    int total_splits = 0;

    h_A_reordered =               (int *)malloc(rows * cols * sizeof(int));
    h_nonzeros_counts =           (int *)malloc(rows * sizeof(int));
    h_nonzeros_counts_reordered = (int *)malloc(rows * sizeof(int));
    h_JDS_index_reordered =       (int *)malloc(rows * sizeof(int));

    hipMalloc((void **)&d_A,                         rows * cols * sizeof(int));
    hipMalloc((void **)&d_A_reordered,               rows * cols * sizeof(int));
    hipMalloc((void **)&d_nonzeros_counts,           rows * sizeof(int));
    hipMalloc((void **)&d_nonzeros_counts_reordered, rows * sizeof(int));
    hipMalloc((void **)&d_A_new_order,               rows * sizeof(int));
    hipMalloc((void **)&d_JDS_index_reordered,       rows * sizeof(int));

    // Count nonzeros per row
    count_nonzeros(h_A, h_nonzeros_counts, rows, cols);

    hipMemcpy(d_A,               h_A,               rows * cols * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_nonzeros_counts, h_nonzeros_counts, rows * sizeof(int),        hipMemcpyHostToDevice);

    // Sort A, JDS_index, nonzeros_counts by nonzeros_counts 
    dim3 dimBlock(BLOCKSIZE);
    dim3 dimGrid((rows + dimBlock.x - 1) / dimBlock.x);
    parallel_count_sort_JDS<<<dimGrid, dimBlock>>>(d_nonzeros_counts, d_nonzeros_counts_reordered,
                                                   d_A_new_order, d_JDS_index_reordered,
                                                   rows, cols);
    parallel_copy<<<dimGrid, dimBlock>>>(d_A_new_order, d_A, d_A_reordered, rows, cols);

    hipMemcpy(h_A_reordered,               d_A_reordered,               rows * cols * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_nonzeros_counts_reordered, d_nonzeros_counts_reordered, rows * sizeof(int),        hipMemcpyDeviceToHost);
    hipMemcpy(h_JDS_index_reordered,       d_JDS_index_reordered,       rows * sizeof(int),        hipMemcpyDeviceToHost);
    
    
    // Split by using the mean of the differences
    splits = (int *)malloc((rows + 1) * sizeof(int));
    total_splits = split_A(splits, h_nonzeros_counts_reordered, rows);
    
    // Use ELL for every split
    hipMalloc((void **)&d_X, cols * sizeof(int));
    hipMemcpy(d_X, h_X, cols * sizeof(int), hipMemcpyHostToDevice);
    for(int i = 0; i < total_splits - 1; ++i){

        int split_rows = 0;
        int split_cols = 0;

        int* h_ELL_data = NULL;
        int* h_ELL_index = NULL;

        int *d_ELL_data = NULL;
        int *d_ELL_data_transposed = NULL;
        int *d_ELL_index = NULL;
        int *d_ELL_index_transposed = NULL;

        int *d_Y = NULL;
        

        split_rows = splits[i + 1] - splits[i];
        split_cols = h_nonzeros_counts_reordered[splits[i]];
        if(split_cols == 0) break;

        h_ELL_data =  (int *)malloc(split_rows * split_cols * sizeof(int));
        h_ELL_index = (int *)malloc(split_rows * split_cols * sizeof(int));

        generate_formats_ELL_COO(h_A_reordered + splits[i] * cols, split_rows, cols, split_cols,
                                 h_ELL_data, h_ELL_index, NULL, NULL, NULL);


        hipMalloc((void **)&d_ELL_data,             split_rows * split_cols * sizeof(int));
        hipMalloc((void **)&d_ELL_data_transposed,  split_rows * split_cols * sizeof(int));
        hipMalloc((void **)&d_ELL_index,            split_rows * split_cols * sizeof(int));
        hipMalloc((void **)&d_ELL_index_transposed, split_rows * split_cols * sizeof(int));

        hipMalloc((void **)&d_Y, split_rows * sizeof(int));

        hipMemcpy(d_ELL_data,  h_ELL_data,  split_rows * split_cols * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_ELL_index, h_ELL_index, split_rows * split_cols * sizeof(int), hipMemcpyHostToDevice);

        
        dim3 dimBlock(TILE_DIM, BLOCK_ROWS);
        dim3 dimGrid((split_rows + dimBlock.x - 1) / dimBlock.x, (split_cols + dimBlock.y - 1) / dimBlock.y);
        transpose<<<dimGrid, dimBlock>>>(d_ELL_data,  d_ELL_data_transposed,  split_rows, split_cols);
        transpose<<<dimGrid, dimBlock>>>(d_ELL_index, d_ELL_index_transposed, split_rows, split_cols);

        dimBlock = dim3(BLOCKSIZE);
        dimGrid = dim3((split_rows + dimBlock.x - 1) / dimBlock.x);
        parallel_ELL<<<dimGrid, dimBlock>>>(d_ELL_data_transposed, d_ELL_index_transposed, d_X, d_Y, split_rows, split_cols);

        for(int j = splits[i]; j < splits[i + 1]; ++j){
            int temp = 0;
            hipMemcpy(&temp, d_Y + j - splits[i], sizeof(int), hipMemcpyDeviceToHost);
            h_Y[h_JDS_index_reordered[j]] = temp;
        }
            

        hipFree(d_ELL_data);
        hipFree(d_ELL_data_transposed);
        hipFree(d_ELL_index);
        hipFree(d_ELL_index_transposed);
        hipFree(d_Y);

        free(h_ELL_data);
        free(h_ELL_index);
    }

    hipFree(d_A);
    hipFree(d_A_reordered);
    hipFree(d_nonzeros_counts);
    hipFree(d_nonzeros_counts_reordered);
    hipFree(d_A_new_order);
    hipFree(d_X);
    hipFree(d_JDS_index_reordered);
    
    free(h_A_reordered);
    free(h_nonzeros_counts);
    free(h_nonzeros_counts_reordered);
    free(splits);
    free(h_JDS_index_reordered);

}

//-------------------------------------------------------------------------------------------------


int count_nonzeros(int *h_A, int *zeros_per_row, int rows, int cols){

    int total_zeros = 0;

    for(int i=0; i<rows; i++)
        zeros_per_row[i] = 0;

    for(int i=0; i<rows; i++){
        for(int j=0; j<cols; j++)
            if(h_A[i*cols+j] == 0)
                ++zeros_per_row[i];
        
        zeros_per_row[i] = cols - zeros_per_row[i];
        total_zeros += zeros_per_row[i];
    }

    return total_zeros;
}

void generate_formats_ELL_COO(int *A, int rows, int cols, int max_nonzeros,
                              int *ELL_data, int *ELL_index,
                              int *COO_data, int *COO_col_index, int *COO_row_index){
    
    int C00_index = 0;
    for(int i = 0; i < rows; ++i){

        int row_nonzeros = 0;
        int current_nonzeros = 0;
        for(int j = 0; j < cols; ++j){
            
            if(A[i * cols + j] != 0){
                
                if(current_nonzeros < max_nonzeros){
                    ELL_data[i * max_nonzeros + row_nonzeros] = A[i * cols + j];
                    ELL_index[i * max_nonzeros + row_nonzeros] = j;
                    ++row_nonzeros;
                }
                else{
                    COO_data[C00_index] = A[i * cols + j];
                    COO_row_index[C00_index] = i;
                    COO_col_index[C00_index] = j;
                    ++C00_index;
                }
                ++current_nonzeros;
            }
        }
    }

}

int split_A(int *splits, int* nonzeros_counts, int rows){

    int *diffs_per_row = (int *)malloc(rows * sizeof(int));
    int total_diffs = 0;
    int mean_diff = 0;
    int split_index = 0;

    diffs_per_row[0] = 0;
    for(int i = 1; i < rows; ++i){
        diffs_per_row[i] = nonzeros_counts[i-1] - nonzeros_counts[i];
        total_diffs += diffs_per_row[i];
    }
    mean_diff = total_diffs / rows;

    split_index = 0;
    splits[split_index] = 0;
    for(int i = 0; i < rows; ++i){
        if(diffs_per_row[i] > mean_diff) 
            splits[++split_index] = i;
    }
    printf("\n");
    splits[++split_index] = rows;

    free(diffs_per_row);

    return split_index + 1;
}


// Original code taken from https://developer.nvidia.com/blog/efficient-matrix-transpose-cuda-cc/
// and modified for arbitrary matrix dimensions
__global__ 
void transpose(const int *idata, int *odata, int rows, int cols)
{
    __shared__ int tile[TILE_DIM][TILE_DIM + 1]; // +1 to avoid bank conflicts

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        if((x < cols) && (y + j < rows))
            tile[threadIdx.y + j][threadIdx.x] = idata[(y + j) * cols + x];

    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        if((x < rows) && (y + j < cols))
            odata[(y + j) * rows + x] = tile[threadIdx.x][threadIdx.y + j];
}


__global__
void parallel_ELL(int *d_ELL_data_transposed, int *d_ELL_index_transposed, 
                  int *d_X, int* d_Y, 
                  int rows, int max_nonzeros){

    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(col < rows) { // The matrix is transposed
        int dot = 0;
        for(int i = 0; i < max_nonzeros; ++i)
            dot += d_ELL_data_transposed[i * rows + col] * d_X[d_ELL_index_transposed[i * rows + col]];
        d_Y[col] = dot;
    }
}


__global__
void parallel_count_sort_JDS(int *d_nonzeros_counts, int *d_nonzeros_counts_reordered,
                             int* d_A_new_order, int *d_JDS_index_reordered,
                             int rows, int cols){

    extern __shared__ int sdata[];

    int tx = threadIdx.x;
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    int nonzeros_count = (row < rows ? d_nonzeros_counts[row] : INT_MIN);
    int my_place = 0;

    int phases = (rows + blockDim.x - 1) / blockDim.x;
    for(int ph = 0; ph < phases; ++ph) {

        if(ph * blockDim.x + tx < rows) sdata[tx] = d_nonzeros_counts[ph * blockDim.x + tx];
        else                            sdata[tx] = INT_MAX;
        __syncthreads();
        
        for (int i = 0; i < blockDim.x; ++i)
            my_place += (nonzeros_count > sdata[i]) || ((nonzeros_count == sdata[i]) && (row < ph * blockDim.x + i));
        __syncthreads();
    }

    if(row < rows){
        d_nonzeros_counts_reordered[rows - 1 - my_place] = nonzeros_count;      // Reverse order
        d_A_new_order[row] =                               rows - 1 - my_place; // Reverse order
        d_JDS_index_reordered[rows - 1 - my_place] =       row;                 // Reverse order
    } 
}


__global__
void parallel_copy(int *d_A_new_order, int *d_A, int *d_A_reordered,
                   int rows, int cols){

    int tx = threadIdx.x;

    for (int i = 0; i < rows; ++i){
        int new_pos = d_A_new_order[i];
        for(int j = tx; j < cols; j += blockDim.x)
            d_A_reordered[new_pos * cols + j] = d_A[i * cols + j];
    }
}