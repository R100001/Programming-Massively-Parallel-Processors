/*/
 *
 *   A simple program to demonstrate array indexing in kernels.
 * 
 *  Compile with:
 *      nvcc array_assign_on_device.cu
 * 
 *  Run with:
 *     ./a.out
 * 
/*/


#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel(int *a, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    a[i] = i;
}

int main()
{

    int N = 4096;
    int threads = 128;
    int blocks = (N + threads - 1) / threads;
    int *a;

    hipMallocManaged(&a, N * sizeof(int));
    kernel<<<blocks, threads>>>(a, N);
    hipDeviceSynchronize();

    for (int i = 0; i < 10; i++)
        printf("%d\n", a[i]);

    hipFree(a);
    return 0;
}
