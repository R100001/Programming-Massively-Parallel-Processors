/*/
 * 
 *  A simple program to test if cuda is working.
 * 
 *  Compile with:
 *      nvcc simple_kernel_launch.cu
 * 
 *  Run with:
 *     ./a.out
 * 
/*/


#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void mykernel(void) {}

int main() {
    mykernel<<<1,1>>>();
    printf("Hello World!\n");
    return 0;
}