#include <stdio.h>

int main()
{
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        printf("%s %.2f\n", prop.name, 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
}