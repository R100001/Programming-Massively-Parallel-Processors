#include "hip/hip_runtime.h"

#include <cstdio>

// computes c(i,j) = a(i,j) + b(i,j)
// In this case j is the fastest changing thread dimension
__global__ void array_add(int *a, int *b, int *c, int m, int n)
{
    
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n && j < m)
    {
        int idx = i * m + j;
        c[idx] = a[idx] + b[idx];
    }
}

void measure_performance(int *d_a, int *d_b, int *d_c,
                         int m, int n,
                         dim3 blocks, dim3 threads){

    hipEvent_t start, stop;
    float milliseconds;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    array_add<<<blocks, threads>>>(d_a, d_b, d_c, m, n);
    hipEventRecord(stop);
    
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);

    int N = m * n;
    int load_write_ops = 3; // load a, load b, write c
    int RW = N * sizeof(int) * load_write_ops;

    printf("\n\n");
    printf("|Metrics:\n");
    printf("|==================================\n");
    printf("|Time elapsed: %f s\n", milliseconds / 1e3);
    printf("|==================================\n");
    printf("|Effective Bandwidth (GB/s): %.3f\n", RW / milliseconds / 1e6);
    printf("|==================================\n");
    printf("\n\n");

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main(int argc, char *argv[])
{

    int m, n, t_y, t_x;
    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;

    if(argc != 5){
        printf("Usage: %s <m> <n> <t_y> <t_x>\n", argv[0]);
        return 1;
    }

    m = atoi(argv[1]);
    n = atoi(argv[2]);
    t_y = atoi(argv[3]);
    t_x = atoi(argv[4]);

    // allocate memory on the host
    h_a = (int*)malloc(m * n * sizeof(int));
    h_b = (int*)malloc(m * n * sizeof(int));
    h_c = (int*)malloc(m * n * sizeof(int));

    // initialize the arrays 'a' and 'b' on the host
    srand(time(NULL));
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++){

            h_a[i * n + j] = rand() % 10;
            h_b[i * n + j] = rand() % 10;
        }

    // allocate memory on the device
    hipMalloc((void**)&d_a, m * n * sizeof(int));
    hipMalloc((void**)&d_b, m * n * sizeof(int));
    hipMalloc((void**)&d_c, m * n * sizeof(int));

    // copy the arrays 'a' and 'b' to the device
    hipMemcpy(d_a, h_a, m * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, m * n * sizeof(int), hipMemcpyHostToDevice);

    dim3 threads(t_x, t_y);
    dim3 blocks((n + threads.x - 1) / threads.x, (m + threads.y - 1) / threads.y);
    measure_performance(d_a, d_b, d_c, m, n, blocks, threads);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);
    
    return 0;
}
