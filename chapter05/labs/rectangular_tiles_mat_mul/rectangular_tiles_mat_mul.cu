#include "hip/hip_runtime.h"
/*/
 * 
 *  This program implements matrix-matrix multiplication.
 *  The implementation uses rectangular tiles to reduce memory usage
 *  loading the elements into shared memory.
 *  Global memory accesses are reduced relatively to the tile dimensions.
 * 
 *  Compile with:
 *      nvcc rectangular_tiles_mat_mul.cu
 * 
 *  Run with:
 *     ./a.out <num_rows_A> <num_columns_A> <num_rows_B> <num_columns_B>
 * 
/*/

#include <stdio.h>

//#define DEBUG

#define TILE_WIDTH 32

// Compute P = M * N
__global__ void matrix_mul(float *M, float *N, float *P,
                           int num_rows_M, int num_cols_M,
                           int num_rows_N, int num_cols_N,
                           int num_rows_P, int num_cols_P) {
    
    // Initialize shared memory
    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH    ];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH * 2];
    
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    int row = by * TILE_WIDTH     + ty;
    int col = bx * TILE_WIDTH * 2 + tx;

    float Pvalue1 = 0;
    float Pvalue2 = 0;
    // Loop over the M and N tiles required to compute the P element
    for (int ph = 0; ph < ceil(num_cols_M/(float)TILE_WIDTH); ph++) {

        // Collaborative loading of M and N tiles into shared memory
        if (row < num_rows_M && ph * TILE_WIDTH + tx < num_cols_M)
                         //M[row][ph * TILE_WIDTH + tx];
            ds_M[ty][tx] = M[row * num_cols_M + ph * TILE_WIDTH + tx];
        else
            ds_M[ty][tx] = 0;

        if (ph * TILE_WIDTH + ty < num_rows_N && col < num_cols_N)
                         //N[ph * TILE_WIDTH + ty][col];
            ds_N[ty][tx] = N[(ph * TILE_WIDTH + ty) * num_cols_N + col];
        else
            ds_N[ty][tx] = 0;

        if(ph * TILE_WIDTH + ty < num_rows_N && TILE_WIDTH + tx < num_cols_N)
                                      //N[ph * TILE_WIDTH + ty][TILE_WIDTH + col];
            ds_N[ty][TILE_WIDTH + tx] = N[(ph * TILE_WIDTH + ty) * num_cols_N + TILE_WIDTH + col];
        else
            ds_N[ty][TILE_WIDTH + tx] = 0;

        // Synchronize to make sure the tiles are loaded
        __syncthreads();

        // Compute the P element
        for (int k = 0; k < TILE_WIDTH; k++){
            Pvalue1 += ds_M[ty][k] * ds_N[k][tx];
            Pvalue2 += ds_M[ty][k] * ds_N[k][TILE_WIDTH + tx];
        }
        
        // Synchronize to make sure the Pvalues are computed
        // before other threads load new tiles
        __syncthreads();
    }

    // Store the Pvalues in P
    if (row < num_rows_P && col < num_cols_P)
      //P[row][col];
        P[row * num_cols_P + col] = Pvalue1;

    if (row < num_rows_P && TILE_WIDTH + col < num_cols_P)
      //P[row][TILE_WIDTH + col];
        P[row * num_cols_P + TILE_WIDTH + col] = Pvalue2;
}


int main(int argc, char **argv) {

    float *hostA, *hostB, *hostC;
    float *deviceA, *deviceB, *deviceC;
    int numARows, numAColumns;
    int numBRows, numBColumns;
    int numCRows, numCColumns;

    if (argc != 5){
        printf("Usage: ./a.out <num_rows_A> <num_columns_A> <num_rows_B> <num_columns_B>\n");
        return 1;
    }

    numARows = atoi(argv[1]);
    numAColumns = atoi(argv[2]);
    numBRows = atoi(argv[3]);
    numBColumns = atoi(argv[4]);

    numCRows = numARows;
    numCColumns = numBColumns;
    
    if(numAColumns != numBRows) {
        printf("Number of columns in A must be the same as the number of rows in B\n");
        return 1;
    }

    // Allocate memory on host
    hostA = (float *) malloc(numARows * numAColumns * sizeof(float));
    hostB = (float *) malloc(numBRows * numBColumns * sizeof(float));
    hostC = (float *) malloc(numCRows * numCColumns * sizeof(float));

    // Allocate memory on device
    hipMalloc((void **) &deviceA, numARows * numAColumns * sizeof(float));
    hipMalloc((void **) &deviceB, numBRows * numBColumns * sizeof(float));
    hipMalloc((void **) &deviceC, numCRows * numCColumns * sizeof(float));

    // Initialize host memory
    srand(time(NULL));
    for (int i = 0; i < numARows; i++)
        for (int j = 0; j < numAColumns; j++)
            hostA[i * numAColumns + j] = rand() / (float) RAND_MAX;

    for (int i = 0; i < numBRows; i++)
        for (int j = 0; j < numBColumns; j++)
            hostB[i * numBColumns + j] = rand() / (float) RAND_MAX;

#ifdef DEBUG
    // Show input matrices
    printf("A:\n");
    for (int i = 0; i < numARows; i++) {
        for (int j = 0; j < numAColumns; j++)
            printf("%f ", hostA[i * numAColumns + j]);
        printf("\n");
    }
    printf("\n");

    printf("B:\n");
    for (int i = 0; i < numBRows; i++) {
        for (int j = 0; j < numBColumns; j++)
            printf("%f ", hostB[i * numBColumns + j]);
        printf("\n");
    }
    printf("\n");
#endif

    // Copy host memory to device
    hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
    dim3 gridDim(ceil(((float)numCColumns / blockDim.x) / 2), ceil((float)numCRows / blockDim.y));
    matrix_mul<<<gridDim, blockDim>>>(deviceA, deviceB, deviceC,
                                      numARows, numAColumns,
                                      numBRows, numBColumns,
                                      numCRows, numCColumns);

    // Copy device memory to host
    hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float), hipMemcpyDeviceToHost);

#ifdef DEBUG
    // Print results
    printf("C:\n");
    for (int i = 0; i < numCRows; i++) {
        for (int j = 0; j < numCColumns; j++)
            printf("%f ", hostC[i * numCColumns + j]);
        printf("\n");
    }
    printf("\n");
#endif

    // Free memory
    free(hostA);
    free(hostB);
    free(hostC);
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    return 0;
}